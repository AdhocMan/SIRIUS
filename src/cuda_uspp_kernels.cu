#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cuda_interface.h"

extern hipStream_t* streams;

__global__ void compute_chebyshev_order1_gpu_kernel
(
    int num_gkvec__,
    double c__,
    double r__,
    hipDoubleComplex* phi0__,
    hipDoubleComplex* phi1__
)
{
    int igk = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockIdx.y;

    if (igk < num_gkvec__)
    {
        int i = array2D_offset(igk, j, num_gkvec__);
        // phi0 * c
        hipDoubleComplex z1 = hipCmul(phi0__[i], make_hipDoubleComplex(c__, 0));
        // phi1 - phi0 * c
        hipDoubleComplex z2 = hipCsub(phi1__[i], z1);
        // (phi1 - phi0 * c) / r
        phi1__[i] = hipCdiv(z2, make_hipDoubleComplex(r__, 0));
    }
}

__global__ void compute_chebyshev_orderk_gpu_kernel
(
    int num_gkvec__,
    double c__,
    double r__,
    hipDoubleComplex* phi0__,
    hipDoubleComplex* phi1__,
    hipDoubleComplex* phi2__
)
{
    int igk = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockIdx.y;

    if (igk < num_gkvec__)
    {
        int i = array2D_offset(igk, j, num_gkvec__);
        // phi1 * c
        hipDoubleComplex z1 = hipCmul(phi1__[i], make_hipDoubleComplex(c__, 0));
        // phi2 - phi1 * c
        hipDoubleComplex z2 = hipCsub(phi2__[i], z1);
        // (phi2 - phi1 * c) * 2 / r
        hipDoubleComplex z3 = hipCmul(z2, make_hipDoubleComplex(2.0 / r__, 0));
        // (phi2 - phi1 * c) * 2 / r - phi0
        phi2__[i] = hipCsub(z3, phi0__[i]);
    }
}

extern "C" void compute_chebyshev_polynomial_gpu(int num_gkvec,
                                                 int n,
                                                 double c,
                                                 double r,
                                                 hipDoubleComplex* phi0,
                                                 hipDoubleComplex* phi1,
                                                 hipDoubleComplex* phi2)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gkvec, grid_t.x), n);

    if (phi2 == NULL)
    {
        compute_chebyshev_order1_gpu_kernel <<<grid_b, grid_t>>>
        (
            num_gkvec,
            c,
            r,
            phi0,
            phi1
        );
    }
    else
    {
        compute_chebyshev_orderk_gpu_kernel <<<grid_b, grid_t>>>
        (
            num_gkvec,
            c,
            r,
            phi0,
            phi1,
            phi2
        );
    }
}


//== #define BLOCK_SIZE 32
//== 
//== __global__ void generate_beta_phi_gpu_kernel(int num_gkvec, 
//==                                              int num_beta,
//==                                              int num_phi,
//==                                              int* beta_t_idx, 
//==                                              double* atom_pos, 
//==                                              double* gkvec, 
//==                                              hipDoubleComplex* beta_pw_type,
//==                                              hipDoubleComplex* phi,
//==                                              hipDoubleComplex* beta_phi)
//== {
//==     int idx_beta = blockDim.x * blockIdx.x + threadIdx.x;
//==     int idx_phi = blockDim.y * blockIdx.y + threadIdx.y;
//==     int ia, offset_t;
//==     double x0, y0, z0;
//== 
//==     if (idx_beta < num_beta)
//==     {
//==         ia = beta_t_idx[array2D_offset(0, idx_beta, 2)];
//==         offset_t = beta_t_idx[array2D_offset(1, idx_beta, 2)];
//==         x0 = atom_pos[array2D_offset(0, ia, 3)];
//==         y0 = atom_pos[array2D_offset(1, ia, 3)];
//==         z0 = atom_pos[array2D_offset(2, ia, 3)];
//==     }
//== 
//==     int N = num_blocks(num_gkvec, BLOCK_SIZE);
//== 
//==     hipDoubleComplex val = make_hipDoubleComplex(0.0, 0.0);
//== 
//==     for (int m = 0; m < N; m++)
//==     {
//==         __shared__ hipDoubleComplex beta_pw_tile[BLOCK_SIZE][BLOCK_SIZE];
//==         __shared__ hipDoubleComplex phi_tile[BLOCK_SIZE][BLOCK_SIZE];
//== 
//==         int bs = (m + 1) * BLOCK_SIZE > num_gkvec ? num_gkvec - m * BLOCK_SIZE : BLOCK_SIZE;
//== 
//==         int igk = m * BLOCK_SIZE + threadIdx.y;
//== 
//==         if (igk < num_gkvec && idx_beta < num_beta)
//==         {
//==             double x1 = gkvec[array2D_offset(igk, 0, num_gkvec)];
//==             double y1 = gkvec[array2D_offset(igk, 1, num_gkvec)];
//==             double z1 = gkvec[array2D_offset(igk, 2, num_gkvec)];
//== 
//==             double p = twopi * (x0 * x1 + y0 * y1 + z0 * z1);
//==             double sinp = sin(p);
//==             double cosp = cos(p);
//== 
//==             beta_pw_tile[threadIdx.x][threadIdx.y] = hipCmul(hipConj(beta_pw_type[array2D_offset(igk, offset_t, num_gkvec)]), 
//==                                                             make_hipDoubleComplex(cosp, sinp));
//== 
//==         }
//==         
//==         igk = m * BLOCK_SIZE + threadIdx.x;
//== 
//==         if (igk < num_gkvec && idx_phi < num_phi)
//==             phi_tile[threadIdx.y][threadIdx.x] = phi[array2D_offset(igk, idx_phi, num_gkvec)];
//== 
//==         __syncthreads();
//== 
//==         for (int i = 0; i < bs; i++) val = hipCadd(val, hipCmul(beta_pw_tile[threadIdx.x][i], phi_tile[threadIdx.y][i]));
//== 
//==         __syncthreads();
//==     }
//== 
//==     if (idx_beta < num_beta && idx_phi < num_phi) beta_phi[array2D_offset(idx_beta, idx_phi, num_beta)] = val;
//== }
//== 
//== 
//== extern "C" void generate_beta_phi_gpu(int num_gkvec, 
//==                                       int num_beta, 
//==                                       int num_phi, 
//==                                       int* beta_t_idx, 
//==                                       double* atom_pos,
//==                                       double* gkvec,
//==                                       void* beta_pw_type,
//==                                       void* phi,
//==                                       void* beta_phi)
//== {
//== 
//==     dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
//==     dim3 numBlocks(num_blocks(num_beta, BLOCK_SIZE), num_blocks(num_phi, BLOCK_SIZE));
//== 
//==     generate_beta_phi_gpu_kernel<<<
//==         numBlocks, 
//==         threadsPerBlock>>>(num_gkvec, 
//==                            num_beta,
//==                            num_phi,
//==                            beta_t_idx, 
//==                            atom_pos,
//==                            gkvec, 
//==                            (hipDoubleComplex*)beta_pw_type,
//==                            (hipDoubleComplex*)phi,
//==                            (hipDoubleComplex*)beta_phi);
//== }

//== __global__ void restore_valence_density_gpu_kernel(int num_gvec_loc,
//==                                                    int* atom_type,
//==                                                    int* num_beta, 
//==                                                    double* atom_pos,
//==                                                    int* gvec,
//==                                                    hipDoubleComplex* pp_complex_density_matrix,
//==                                                    int ldm,
//==                                                    hipDoubleComplex** q_pw,
//==                                                    hipDoubleComplex* f_pw)
//== {
//==     extern __shared__ char sdata_ptr[];
//==     hipDoubleComplex* sdata = (hipDoubleComplex*)&sdata_ptr[0];
//== 
//==     int ia = blockIdx.x;
//== 
//==     int iat = atom_type[ia];
//== 
//==     int nbf = num_beta[iat];
//== 
//==     hipDoubleComplex* q_pw_t = q_pw[iat];
//==     //printf("ia : %i, type : %i, nbf : %i, q_pw : %p", ia, iat, nbf, q_pw_t);
//== 
//==     double ax = atom_pos[array2D_offset(0, ia, 3)];
//==     double ay = atom_pos[array2D_offset(1, ia, 3)];
//==     double az = atom_pos[array2D_offset(2, ia, 3)];
//== 
//==     if (threadIdx.x == 0)
//==     {
//==         for (int xi2 = 0; xi2 < nbf; xi2++)
//==         {
//==             for (int xi1 = 0; xi1 <= xi2; xi1++)
//==             {
//==                 int idx12 = xi2 * (xi2 + 1) / 2 + xi1;
//==                 sdata[idx12] = pp_complex_density_matrix[array4D_offset(xi2, xi1, 0, ia, ldm, ldm, 1)];
//==             }
//==         }
//==     }
//==     __syncthreads();
//== 
//==     hipDoubleComplex* f_pw_a = &f_pw[array2D_offset(0, ia, num_gvec_loc)];
//==     
//==     int N = num_blocks(num_gvec_loc, blockDim.x);
//== 
//==     for (int n = 0; n < N; n++)
//==     {
//==         int igloc = n * blockDim.x + threadIdx.x;
//==         if (igloc < num_gvec_loc)
//==         {
//==             int gvx = gvec[array2D_offset(0, igloc, 3)];
//==             int gvy = gvec[array2D_offset(1, igloc, 3)];
//==             int gvz = gvec[array2D_offset(2, igloc, 3)];
//== 
//==             double p = twopi * (ax * gvx + ay * gvy + az * gvz);
//==             
//==             double sinp = sin(p);
//==             double cosp = cos(p);
//== 
//==             hipDoubleComplex zval = make_hipDoubleComplex(0.0, 0.0);
//== 
//==             // \sum_{xi1, xi2} D_{xi2,xi1} * Q(G)_{xi1, xi2}
//==             for (int xi2 = 0; xi2 < nbf; xi2++)
//==             {
//==                 int idx12 = xi2 * (xi2 + 1) / 2;
//== 
//==                 //hipDoubleComplex q = hipCmul(make_hipDoubleComplex(cosp, -sinp), q_pw_t[array2D_offset(igloc, idx12 + xi2, num_gvec_loc)]);
//== 
//==                 // add diagonal term
//==                 //f_pw_a[igloc] = hipCadd(f_pw_a[igloc], hipCmul(sdata[idx12 + xi2], q));
//==                 zval = hipCadd(zval, hipCmul(sdata[idx12 + xi2], q_pw_t[array2D_offset(igloc, idx12 + xi2, num_gvec_loc)]));
//== 
//==                 // add non-diagonal terms
//==                 for (int xi1 = 0; xi1 < xi2; xi1++, idx12++)
//==                 {
//==                     hipDoubleComplex q = q_pw_t[array2D_offset(igloc, idx12, num_gvec_loc)];
//==                     //q = hipCmul(make_hipDoubleComplex(cosp, -sinp), q_pw_t[array2D_offset(igloc, idx12, num_gvec_loc)]);
//==                     
//==                     //double d = 2 * hipCreal(hipCmul(sdata[idx12], q));
//== 
//==                     //f_pw_a[igloc] = hipCadd(f_pw_a[igloc], make_hipDoubleComplex(d, 0));
//==                     //double d = 2 * hipCreal(hipCmul(sdata[idx12], q_pw_t[array2D_offset(igloc, idx12, num_gvec_loc)])
//==                     zval.x += 2 * (sdata[idx12].x * q.x - sdata[idx12].y * q.y);
//==                     //zval = hipCadd(zval, make_hipDoubleComplex(2 * hipCreal(hipCmul(sdata[idx12], q_pw_t[array2D_offset(igloc, idx12, num_gvec_loc)])), 0.0));
//==                 }
//==             }
//==             f_pw_a[igloc] = hipCadd(f_pw_a[igloc], hipCmul(zval, make_hipDoubleComplex(cosp, -sinp))); 
//==         }
//==     }
//== }
//== 
//== __global__ void reduce_rho_pw_kernel(int num_atoms, int num_gvec_loc, hipDoubleComplex* f_pw, hipDoubleComplex* rho_pw)
//== {
//==     int igloc = blockDim.x * blockIdx.x + threadIdx.x;
//== 
//==     if (igloc < num_gvec_loc)
//==     {
//==         for (int ia = 0; ia < num_atoms; ia++) 
//==             rho_pw[igloc] = hipCadd(rho_pw[igloc], f_pw[array2D_offset(igloc, ia, num_gvec_loc)]);
//==     }
//== }
//== 
//== 
//== extern "C" void restore_valence_density_gpu(int num_atoms, 
//==                                             int num_gvec_loc,
//==                                             int* atom_type,
//==                                             int* num_beta, 
//==                                             double* atom_pos, 
//==                                             int* gvec,
//==                                             void* pp_complex_density_matrix,
//==                                             int ldm,
//==                                             void** q_pw,
//==                                             void* rho_pw)
//== {
//==     dim3 threadsPerBlock(1024);
//==     dim3 numBlocks(num_atoms);
//== 
//==     hipDoubleComplex* f_pw;
//==     f_pw = (hipDoubleComplex*)cuda_malloc(num_gvec_loc * num_atoms * sizeof(hipDoubleComplex));
//==     cuda_memset(f_pw, 0, num_gvec_loc * num_atoms * sizeof(hipDoubleComplex));
//== 
//==     restore_valence_density_gpu_kernel<<<
//==         numBlocks,
//==         threadsPerBlock,
//==         sizeof(hipDoubleComplex) * ldm * (ldm + 1) / 2>>>(num_gvec_loc,
//==                                                          atom_type,
//==                                                          num_beta, 
//==                                                          atom_pos, 
//==                                                          gvec, 
//==                                                          (hipDoubleComplex*)pp_complex_density_matrix,
//==                                                          ldm,
//==                                                          (hipDoubleComplex**)q_pw,
//==                                                          f_pw);
//==     
//==     cuda_memset(rho_pw, 0, num_gvec_loc * sizeof(hipDoubleComplex));
//==     
//==     dim3 grid_t(128);
//==     dim3 grid_b(num_blocks(num_gvec_loc, grid_t.x));
//==     reduce_rho_pw_kernel<<<grid_b, grid_t>>>
//==         (num_atoms, num_gvec_loc, f_pw, (hipDoubleComplex*)rho_pw);
//==     
//==     cuda_device_synchronize();
//==     cuda_free(f_pw);
//== }




//== __global__ void restore_valence_density_gpu_kernel_v2
//== (
//==     int num_gvec_loc,
//==     int num_beta, 
//==     double ax,
//==     double ay,
//==     double az,
//==     int* gvec,
//==     hipDoubleComplex* pp_complex_density_matrix,
//==     int ldm,
//==     hipDoubleComplex* q_pw_t,
//==     hipDoubleComplex* rho_pw
//== )
//== {
//==     extern __shared__ char sdata_ptr[];
//==     hipDoubleComplex* sdata = (hipDoubleComplex*)&sdata_ptr[0];
//== 
//==     if (threadIdx.x == 0)
//==     {
//==         for (int xi2 = 0; xi2 < num_beta; xi2++)
//==         {
//==             for (int xi1 = 0; xi1 <= xi2; xi1++)
//==             {
//==                 int idx12 = xi2 * (xi2 + 1) / 2 + xi1;
//==                 sdata[idx12] = pp_complex_density_matrix[array3D_offset(xi2, xi1, 0, ldm, ldm)];
//==             }
//==         }
//==     }
//==     __syncthreads();
//== 
//==     int igloc = blockIdx.x * blockDim.x + threadIdx.x;
//==     if (igloc < num_gvec_loc)
//==     {
//==         int gvx = gvec[array2D_offset(0, igloc, 3)];
//==         int gvy = gvec[array2D_offset(1, igloc, 3)];
//==         int gvz = gvec[array2D_offset(2, igloc, 3)];
//== 
//==         double p = twopi * (ax * gvx + ay * gvy + az * gvz);
//==         
//==         double sinp = sin(p);
//==         double cosp = cos(p);
//== 
//==         hipDoubleComplex zval = make_hipDoubleComplex(0.0, 0.0);
//== 
//==         // \sum_{xi1, xi2} D_{xi2,xi1} * Q(G)_{xi1, xi2}
//==         for (int xi2 = 0; xi2 < num_beta; xi2++)
//==         {
//==             int idx12 = xi2 * (xi2 + 1) / 2;
//== 
//==             // add diagonal term
//==             zval = hipCadd(zval, hipCmul(sdata[idx12 + xi2], q_pw_t[array2D_offset(igloc, idx12 + xi2, num_gvec_loc)]));
//== 
//==             // add non-diagonal terms
//==             for (int xi1 = 0; xi1 < xi2; xi1++, idx12++)
//==             {
//==                 hipDoubleComplex q = q_pw_t[array2D_offset(igloc, idx12, num_gvec_loc)];
//==                 zval.x += 2 * (sdata[idx12].x * q.x - sdata[idx12].y * q.y);
//==             }
//==         }
//==         rho_pw[igloc] = hipCadd(rho_pw[igloc], hipCmul(zval, make_hipDoubleComplex(cosp, -sinp))); 
//==     }
//== }
//== 
//== extern "C" void restore_valence_density_gpu_v2(int num_gvec_loc,
//==                                                int num_beta,
//==                                                double ax,
//==                                                double ay,
//==                                                double az,
//==                                                int* gvec,
//==                                                void* pp_complex_density_matrix,
//==                                                int ldm,
//==                                                void* q_pw_t,
//==                                                void* rho_pw,
//==                                                int stream_id)
//== {
//==     hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];
//== 
//==     dim3 grid_t(64);
//==     dim3 grid_b(num_blocks(num_gvec_loc, grid_t.x));
//== 
//==     restore_valence_density_gpu_kernel_v2<<<grid_b, grid_t, sizeof(hipDoubleComplex) * ldm * (ldm + 1) / 2, stream>>>
//==         (num_gvec_loc, num_beta, ax, ay, az, gvec, (hipDoubleComplex*)pp_complex_density_matrix, ldm,
//==          (hipDoubleComplex*)q_pw_t, (hipDoubleComplex*)rho_pw);
//== }

__global__ void mul_veff_with_phase_factors_gpu_kernel(int num_gvec_loc__,
                                                       hipDoubleComplex const* veff__, 
                                                       int const* gvec__, 
                                                       double const* atom_pos__, 
                                                       hipDoubleComplex* veff_a__)
{
    int igloc = blockDim.x * blockIdx.x + threadIdx.x;
    int ia = blockIdx.y;

    if (igloc < num_gvec_loc__)
    {
        int gvx = gvec__[array2D_offset(0, igloc, 3)];
        int gvy = gvec__[array2D_offset(1, igloc, 3)];
        int gvz = gvec__[array2D_offset(2, igloc, 3)];
        double ax = atom_pos__[array2D_offset(0, ia, 3)];
        double ay = atom_pos__[array2D_offset(1, ia, 3)];
        double az = atom_pos__[array2D_offset(2, ia, 3)];

        double p = twopi * (ax * gvx + ay * gvy + az * gvz);
            
        veff_a__[array2D_offset(igloc, ia, num_gvec_loc__)] = hipCmul(veff__[igloc], make_hipDoubleComplex(cos(p), -sin(p)));
    }
}
 
extern "C" void mul_veff_with_phase_factors_gpu(int num_atoms__,
                                                int num_gvec_loc__, 
                                                hipDoubleComplex const* veff__, 
                                                int const* gvec__, 
                                                double const* atom_pos__,
                                                hipDoubleComplex* veff_a__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec_loc__, grid_t.x), num_atoms__);

    mul_veff_with_phase_factors_gpu_kernel <<<grid_b, grid_t>>>
    (
        num_gvec_loc__,
        veff__,
        gvec__,
        atom_pos__,
        veff_a__
    );
}
//==__global__ void compute_d_mtrx_gpu_kernel
//==(
//==    int num_gvec_loc, 
//==    hipDoubleComplex const* vtmp, 
//==    hipDoubleComplex const* q_pw_t, 
//==    hipDoubleComplex* d_mtrx_gpu
//==)
//=={
//==    int idx = blockIdx.x;
//==
//==    //int N = num_blocks(num_gvec_loc, blockDim.x);
//==
//==    extern __shared__ char sdata_ptr[];
//==    hipDoubleComplex* sdata = (hipDoubleComplex*)&sdata_ptr[0];
//==
//==    //sdata[threadIdx.x] = make_hipDoubleComplex(0.0, 0.0);
//==
//==    hipDoubleComplex z = make_hipDoubleComplex(0, 0);
//==    int igloc = threadIdx.x;
//==    while (igloc < num_gvec_loc)
//==    {
//==        z = hipCadd(z, hipCmul(vtmp[igloc], hipConj(q_pw_t[array2D_offset(igloc, idx,  num_gvec_loc)])));
//==        igloc += blockDim.x;
//==    }
//==    sdata[threadIdx.x] = z;
//==    __syncthreads();
//==
//==    //== for (int n = 0; n < N; n++)
//==    //== {
//==    //==     int igloc = n * blockDim.x + threadIdx.x;
//==    //==     if (igloc < num_gvec_loc)
//==    //==     {
//==    //==         sdata[threadIdx.x] = hipCadd(sdata[threadIdx.x], 
//==    //==                                     hipCmul(vtmp[igloc], 
//==    //==                                            hipConj(q_pw_t[array2D_offset(igloc, idx,  num_gvec_loc)])));
//==    //==     }
//==    //== }
//==    //== 
//==    //== __syncthreads();
//==
//==    for (int s = 1; s < blockDim.x; s *= 2) 
//==    {
//==        if (threadIdx.x % (2 * s) == 0) sdata[threadIdx.x] = hipCadd(sdata[threadIdx.x], sdata[threadIdx.x + s]);
//==        __syncthreads();
//==    }
//==
//==    d_mtrx_gpu[idx] = sdata[0];
//==}

//== extern "C" void compute_d_mtrx_valence_gpu(int num_gvec_loc,
//==                                            int num_elements,
//==                                            hipDoubleComplex const* veff, 
//==                                            int const* gvec, 
//==                                            double ax,
//==                                            double ay,
//==                                            double az,
//==                                            hipDoubleComplex* vtmp,
//==                                            hipDoubleComplex const* q_pw_t,
//==                                            hipDoubleComplex* d_mtrx,
//==                                            int stream_id)
//== {
//==     hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];
//== 
//==     dim3 grid_t(128);
//== 
//==     dim3 grid_b(num_blocks(num_gvec_loc, grid_t.x));
//== 
//==     mul_veff_with_phase_factors_gpu_kernel <<<grid_b, grid_t, 0, stream>>>
//==     (
//==         num_gvec_loc,
//==         veff,
//==         gvec,
//==         ax,
//==         ay,
//==         az, 
//==         vtmp
//==     );
//== 
//==     grid_b = dim3(num_elements);
//== 
//==     compute_d_mtrx_gpu_kernel <<<grid_b, grid_t, grid_t.x * sizeof(hipDoubleComplex), stream>>>
//==     (
//==         num_gvec_loc,
//==         vtmp,
//==         q_pw_t,
//==         d_mtrx
//==     );
//== 
//== }



// TODO: proper order of indices for atom_pos

__global__ void generate_phase_factors_conj_gpu_kernel
(
    int num_gvec_loc, 
    int num_atoms, 
    double* atom_pos, 
    int* gvec, 
    hipDoubleComplex* phase_factors
)
{
    int ia = blockIdx.y;
    int igloc = blockIdx.x * blockDim.x + threadIdx.x;

    if (igloc < num_gvec_loc)
    {
        int gvx = gvec[array2D_offset(0, igloc, 3)];
        int gvy = gvec[array2D_offset(1, igloc, 3)];
        int gvz = gvec[array2D_offset(2, igloc, 3)];
    
        double ax = atom_pos[array2D_offset(ia, 0, num_atoms)];
        double ay = atom_pos[array2D_offset(ia, 1, num_atoms)];
        double az = atom_pos[array2D_offset(ia, 2, num_atoms)];

        double p = twopi * (ax * gvx + ay * gvy + az * gvz);

        double sinp = sin(p);
        double cosp = cos(p);

        phase_factors[array2D_offset(igloc, ia, num_gvec_loc)] = make_hipDoubleComplex(cosp, -sinp);
    }
}

__global__ void generate_phase_factors_gpu_kernel
(
    int num_gvec_loc, 
    double const* atom_pos, 
    int const* gvec, 
    hipDoubleComplex* phase_factors
)
{
    int ia = blockIdx.y;
    int igloc = blockIdx.x * blockDim.x + threadIdx.x;

    if (igloc < num_gvec_loc)
    {
        int gvx = gvec[array2D_offset(0, igloc, 3)];
        int gvy = gvec[array2D_offset(1, igloc, 3)];
        int gvz = gvec[array2D_offset(2, igloc, 3)];
    
        double ax = atom_pos[array2D_offset(0, ia, 3)];
        double ay = atom_pos[array2D_offset(1, ia, 3)];
        double az = atom_pos[array2D_offset(2, ia, 3)];

        double p = twopi * (ax * gvx + ay * gvy + az * gvz);

        double sinp = sin(p);
        double cosp = cos(p);

        phase_factors[array2D_offset(igloc, ia, num_gvec_loc)] = make_hipDoubleComplex(cosp, sinp);
    }
}


extern "C" void generate_d_mtrx_pw_gpu(int num_atoms,
                                       int num_gvec_loc,
                                       int num_beta,
                                       double* atom_pos,
                                       int* gvec,
                                       hipDoubleComplex* d_mtrx_packed,
                                       hipDoubleComplex* d_mtrx_pw)
{
    CUDA_timer t("generate_d_mtrx_pw_gpu");

    hipDoubleComplex* phase_factors;
    phase_factors = (hipDoubleComplex*)cuda_malloc(num_gvec_loc * num_atoms * sizeof (hipDoubleComplex));

    dim3 grid_t(32);
    dim3 grid_b(num_blocks(num_gvec_loc, grid_t.x), num_atoms);

    generate_phase_factors_conj_gpu_kernel<<<grid_b, grid_t>>>
    (
        num_gvec_loc, 
        num_atoms, 
        atom_pos, 
        gvec, 
        phase_factors
    );
    
    hipDoubleComplex zone = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex zzero = make_hipDoubleComplex(0.0, 0.0);

    cublas_zgemm(0, 1, num_gvec_loc, num_beta * num_beta, num_atoms, &zone, 
                 phase_factors, num_gvec_loc, d_mtrx_packed, num_beta * num_beta, &zzero,
                 d_mtrx_pw, num_gvec_loc, -1);

    cuda_free(phase_factors);
}

extern "C" void generate_phase_factors_gpu(int num_gvec_loc__,
                                           int num_atoms__,
                                           int const* gvec__,
                                           double const* atom_pos__,
                                           hipDoubleComplex* phase_factors__)

{
    dim3 grid_t(32);
    dim3 grid_b(num_blocks(num_gvec_loc__, grid_t.x), num_atoms__);

    generate_phase_factors_gpu_kernel<<<grid_b, grid_t>>>
    (
        num_gvec_loc__, 
        atom_pos__, 
        gvec__, 
        phase_factors__
    );
}

__global__ void sum_q_pw_d_mtrx_pw_gpu_kernel
(
    int num_gvec_loc,
    int num_beta,
    hipDoubleComplex* q_pw_t,
    hipDoubleComplex* d_mtrx_pw,
    hipDoubleComplex* rho_pw
)
{
    int igloc = blockIdx.x * blockDim.x + threadIdx.x;
    if (igloc < num_gvec_loc)
    {
        hipDoubleComplex zval = make_hipDoubleComplex(0.0, 0.0);

        // \sum_{xi1, xi2} D_{xi2,xi1} * Q(G)_{xi1, xi2}
        for (int xi2 = 0; xi2 < num_beta; xi2++)
        {
            int idx12 = xi2 * (xi2 + 1) / 2;

            // add diagonal term
            zval = hipCadd(zval, hipCmul(d_mtrx_pw[array2D_offset(igloc, xi2 * num_beta + xi2, num_gvec_loc)], 
                                       q_pw_t[array2D_offset(igloc, idx12 + xi2, num_gvec_loc)]));

            // add non-diagonal terms
            for (int xi1 = 0; xi1 < xi2; xi1++, idx12++)
            {
                hipDoubleComplex q = q_pw_t[array2D_offset(igloc, idx12, num_gvec_loc)];
                hipDoubleComplex d1 = d_mtrx_pw[array2D_offset(igloc, xi2 * num_beta + xi1, num_gvec_loc)];
                hipDoubleComplex d2 = d_mtrx_pw[array2D_offset(igloc, xi1 * num_beta + xi2, num_gvec_loc)];

                zval = hipCadd(zval, hipCmul(q, d1));
                zval = hipCadd(zval, hipCmul(hipConj(q), d2));
            }
        }
        rho_pw[igloc] = hipCadd(rho_pw[igloc], zval);
    }
}

extern "C" void sum_q_pw_d_mtrx_pw_gpu(int num_gvec_loc,
                                       int num_beta,
                                       hipDoubleComplex* q_pw_t,
                                       hipDoubleComplex* d_mtrx_pw,
                                       hipDoubleComplex* rho_pw)
{
    CUDA_timer t("sum_q_pw_d_mtrx_pw_gpu");

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec_loc, grid_t.x));
    
    sum_q_pw_d_mtrx_pw_gpu_kernel <<<grid_b, grid_t>>>
    (
        num_gvec_loc, 
        num_beta, 
        q_pw_t, 
        d_mtrx_pw, 
        rho_pw
    );
}

__global__ void copy_beta_psi_gpu_kernel
(
    hipDoubleComplex const* beta_psi,
    int beta_psi_ld, 
    double const* wo,
    hipDoubleComplex* beta_psi_wo,
    int beta_psi_wo_ld
)
{
    int xi = threadIdx.x;
    int j = blockIdx.x;

    beta_psi_wo[array2D_offset(xi, j, beta_psi_wo_ld)] = hipCmul(hipConj(beta_psi[array2D_offset(xi, j, beta_psi_ld)]),
                                                                make_hipDoubleComplex(wo[j], 0.0));
}

extern "C" void copy_beta_psi_gpu(int nbf,
                                  int nloc,
                                  hipDoubleComplex const* beta_psi,
                                  int beta_psi_ld,
                                  double const* wo,
                                  hipDoubleComplex* beta_psi_wo,
                                  int beta_psi_wo_ld,
                                  int stream_id)
{
    dim3 grid_t(nbf);
    dim3 grid_b(nloc);
    
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];
    
    copy_beta_psi_gpu_kernel <<<grid_b, grid_t, 0, stream>>>
    (
        beta_psi,
        beta_psi_ld,
        wo,
        beta_psi_wo,
        beta_psi_wo_ld
    );
}

__global__ void compute_inner_product_gpu_kernel
(
    int num_gkvec_row,
    hipDoubleComplex const* f1,
    hipDoubleComplex const* f2,
    double* prod
)
{
    int N = num_blocks(num_gkvec_row, blockDim.x);

    extern __shared__ char sdata_ptr[];
    double* sdata = (double*)&sdata_ptr[0];

    sdata[threadIdx.x] = 0.0;

    for (int n = 0; n < N; n++)
    {
        int igk = n * blockDim.x + threadIdx.x;
        if (igk < num_gkvec_row)
        {
            int k = array2D_offset(igk, blockIdx.x, num_gkvec_row);
            sdata[threadIdx.x] += f1[k].x * f2[k].x + f1[k].y *f2[k].y;
        }
    }

    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) 
    {
        if (threadIdx.x % (2 * s) == 0) sdata[threadIdx.x] = sdata[threadIdx.x] + sdata[threadIdx.x + s];
        __syncthreads();
    }
    
    prod[blockIdx.x] = sdata[0];
}

extern "C" void compute_inner_product_gpu(int num_gkvec_row,
                                          int n,
                                          hipDoubleComplex const* f1,
                                          hipDoubleComplex const* f2,
                                          double* prod)
{
    dim3 grid_t(64);
    dim3 grid_b(n);

    compute_inner_product_gpu_kernel <<<grid_b, grid_t, grid_t.x * sizeof(double)>>>
    (
        num_gkvec_row,
        f1,
        f2,
        prod
    );
}


