#include "hip/hip_runtime.h"
// This file must be compiled with nvcc

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

hipblasHandle_t& cublas_handle()
{
    static hipblasHandle_t handle;
    static bool init = false;

    if (!init)
    {
        if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
        {
            printf("hipblasCreate() failed \n");
            exit(-1);
        }
        init = true;
    }
    
    return handle;
}

/*extern "C" void cuda_init()
{
    if (hipInit(0) != hipSuccess)
    {
        printf("hipInit failed\n");
    }
}*/

extern "C" void cublas_init()
{
    cublas_handle();
}

extern "C" void cuda_malloc_host(void** ptr, size_t size)
{
    if (hipHostMalloc(ptr, size) != hipSuccess)
    {  
        printf("hipHostMalloc failed\n");
        exit(-1);
    }
}

extern "C" void cuda_free_host(void** ptr)
{
    if (hipHostFree(*ptr) != hipSuccess)
    {
        printf("hipHostFree failed\n");
        exit(-1);
    }
}

extern "C" void cuda_malloc(void **ptr, size_t size)
{
    if (hipMalloc(ptr, size) != hipSuccess)
    {
        printf("failed to execute hipMalloc() \n");
        exit(0);
    }
}

extern "C" void cuda_free(void *ptr)
{
    if (hipFree(ptr) != hipSuccess)
    {
        printf("failed to execute hipFree() \n");
        exit(0);
    }
}

extern "C" void cuda_copy_to_device(void *target, void *source, size_t size)
{
    if (hipMemcpy(target, source, size, hipMemcpyHostToDevice) != hipSuccess)
    {
        printf("failed to execute hipMemcpy(hipMemcpyHostToDevice)\n");
        exit(0);
    }
}

extern "C" void cuda_copy_to_host(void *target, void *source, size_t size)
{
    if (hipMemcpy(target, source, size, hipMemcpyDeviceToHost) != hipSuccess)
    {
        printf("failed to execute hipMemcpy(hipMemcpyDeviceToHost)\n");
        exit(0);
    }
}

hipStream_t* streams;

extern "C" void cuda_create_streams(int num_streams)
{
    streams = (hipStream_t*)malloc(num_streams * sizeof(hipStream_t));
    for (int i = 0; i < num_streams; i++) hipStreamCreate(&streams[i]);
}

extern "C" void cuda_destroy_streams(int num_streams)
{
    for (int i = 0; i < num_streams; i++) hipStreamDestroy(streams[i]);
    free(streams);
}

extern "C" void cuda_stream_synchronize(int stream_id)
{
    if (hipStreamSynchronize(streams[stream_id]) != hipSuccess)
    {
        printf("failed to execute hipStreamSynchronize()\n");
        exit(0);
    }
}

extern "C" void cuda_async_copy_to_device(void *target, void *source, size_t size, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];

    if (hipMemcpyAsync(target, source, size, hipMemcpyHostToDevice, stream) != hipSuccess)
    {
        printf("failed to execute hipMemcpy(hipMemcpyHostToDevice)\n");
        exit(0);
    }
}

extern "C" void cuda_async_copy_to_host(void *target, void *source, size_t size, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];

    if (hipMemcpyAsync(target, source, size, hipMemcpyDeviceToHost, stream) != hipSuccess)
    {
        printf("failed to execute hipMemcpy(hipMemcpyDeviceToHost)\n");
        exit(0);
    }
}

extern "C" void cuda_memset(void *ptr,int value, size_t size)
{
    if (hipMemset(ptr, value, size) != hipSuccess)
    {
        printf("failed to execute hipMemset()\n");
        exit(0);
    }
}

extern "C" void cuda_host_register(void* ptr, size_t size)
{
    assert(ptr);
    
    hipError_t err = hipHostRegister(ptr, size, 0);
    if (err != hipSuccess)
    {
        printf("failed to execute hipHostRegister\n");
        switch (err)
        {
            case hipErrorInvalidValue:
                printf("hipErrorInvalidValue\n");
                break;
            case hipErrorOutOfMemory:
                printf("hipErrorOutOfMemory\n");
                break;
            default:
                printf("unrecognized error\n");
        }
        exit(-1);
    }
}

extern "C" void cuda_host_unregister(void* ptr)
{
    if (hipHostUnregister(ptr) != hipSuccess)
    {
        printf("failed to execute hipHostUnregister\n");
        exit(-1);
    }
}

//* hipDeviceProp_t& cuda_devprop()
//* {
//*     static hipDeviceProp_t devprop;
//* 
//*     return devprop;
//* }

extern "C" void cuda_device_info()
{
    int count;
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("failed to execute hipGetDeviceCount() \n");
        exit(-1);
    }

    if (count == 0)
    {
        printf("no avaiable devices\n");
        exit(-1);
    }

    hipDeviceProp_t devprop;
     
    if (hipGetDeviceProperties(&devprop, 0) != hipSuccess)
    {
        printf("failed to execute hipGetDeviceProperties()\n");
        exit(-1);
    }
    
    printf("name                        : %s \n", devprop.name);
    printf("major                       : %i \n", devprop.major);
    printf("minor                       : %i \n", devprop.minor);
    printf("asyncEngineCount            : %i \n", devprop.asyncEngineCount);
    printf("canMapHostMemory            : %i \n", devprop.canMapHostMemory);
    printf("clockRate                   : %i kHz \n", devprop.clockRate);
    printf("concurrentKernels           : %i \n", devprop.concurrentKernels);
    printf("ECCEnabled                  : %i \n", devprop.ECCEnabled);
    printf("l2CacheSize                 : %i kB \n", devprop.l2CacheSize/1024);
    printf("maxGridSize                 : %i %i %i \n", devprop.maxGridSize[0], devprop.maxGridSize[1], devprop.maxGridSize[2]);
    printf("maxThreadsDim               : %i %i %i \n", devprop.maxThreadsDim[0], devprop.maxThreadsDim[1], devprop.maxThreadsDim[2]);
    printf("maxThreadsPerBlock          : %i \n", devprop.maxThreadsPerBlock);
    printf("maxThreadsPerMultiProcessor : %i \n", devprop.maxThreadsPerMultiProcessor);
    printf("memoryBusWidth              : %i bits \n", devprop.memoryBusWidth);
    printf("memoryClockRate             : %i kHz \n", devprop.memoryClockRate);
    printf("memPitch                    : %zi \n", devprop.memPitch);
    printf("multiProcessorCount         : %i \n", devprop.multiProcessorCount);
    printf("regsPerBlock                : %i \n", devprop.regsPerBlock);
    printf("sharedMemPerBlock           : %li kB \n", devprop.sharedMemPerBlock/1024);
    printf("totalConstMem               : %li kB \n", devprop.totalConstMem/1024);
    printf("totalGlobalMem              : %li kB \n", devprop.totalGlobalMem/1024);
}


extern "C" void cublas_zgemm(int transa, int transb, int32_t m, int32_t n, int32_t k, 
                             void* alpha, void* a, int32_t lda, void* b, 
                             int32_t ldb, void* beta, void* c, int32_t ldc)
{
    const hipblasOperation_t trans[] = {HIPBLAS_OP_N, HIPBLAS_OP_T, HIPBLAS_OP_C};

    if (hipblasZgemm(cublas_handle(), trans[transa], trans[transb], m, n, k, (hipDoubleComplex*)alpha, (hipDoubleComplex*)a, lda, 
                   (hipDoubleComplex*)b, ldb, (hipDoubleComplex*)beta, (hipDoubleComplex*)c, ldc) != HIPBLAS_STATUS_SUCCESS)
    {
        printf("failed to execute hipblasZgemm() \n");
        exit(-1);
    }
}

// A(GPU) => B(CPU)
extern "C" void cublas_get_matrix(int rows, int cols, int elemSize, const void *A, int lda, void *B, int ldb)
{
    if (hipblasGetMatrix(rows, cols, elemSize, A, lda, B, ldb) != HIPBLAS_STATUS_SUCCESS)
    {
        printf("failed to execute hipblasGetMatrix\n");
        exit(-1);
    }
}

// A(CPU) => B(GPU)
extern "C" void cublas_set_matrix(int rows, int cols, int elemSize, const void *A, int lda, void *B, int ldb)
{
    if (hipblasSetMatrix(rows, cols, elemSize, A, lda, B, ldb) != HIPBLAS_STATUS_SUCCESS)
    {
        printf("failed to execute hipblasSetMatrix\n");
        exit(-1);
    }
}


__device__ size_t array2D_offset(int i0, int i1, int ld0)
{
    return i0 + i1 * ld0;
}

__device__ size_t array3D_offset(int i0, int i1, int i2, int ld0, int ld1)
{
    return i0 + i1 * ld0 + i2 * ld0 * ld1;
}

__device__ size_t array4D_offset(int i0, int i1, int i2, int i3, int ld0, int ld1, int ld2)
{
    return i0 + i1 * ld0 + i2 * ld0 * ld1 + i3 * ld0 * ld1 * ld2;
}

template <typename T, typename U>
__device__ U spline_inner_product_gpu_function(int ld, int size, double* r_dr, T* s1_coefs, U* s2_coefs)
{
    int N = size / blockDim.x;
    if (size % blockDim.x != 0) N++;

    extern __shared__ char sdata_ptr[];
    U* sdata = (U*)&sdata_ptr[0];

    int a_offs = 0 * ld;
    int b_offs = 1 * ld;
    int c_offs = 2 * ld;
    int d_offs = 3 * ld;

    sdata[threadIdx.x] = 0;

    for (int n = 0; n < N; n++)
    {
        int i = n * blockDim.x + threadIdx.x;
        if (i < size - 1)
        {
            double x0 = r_dr[i];
            double dx = r_dr[ld + i];

            T a1 = s1_coefs[a_offs + i];
            T b1 = s1_coefs[b_offs + i];
            T c1 = s1_coefs[c_offs + i];
            T d1 = s1_coefs[d_offs + i];
            
            U a2 = s2_coefs[a_offs + i];
            U b2 = s2_coefs[b_offs + i];
            U c2 = s2_coefs[c_offs + i];
            U d2 = s2_coefs[d_offs + i];
                
            U a1a2 = a1 * a2;
            U d1d2 = d1 * d2;
                
            U k1 = d1 * b2 + c1 * c2 + b1 * d2;

            U k2 = d1 * a2 + c1 * b2 + b1 * c2 + a1 * d2;

            U k3 = c1 * a2 + b1 * b2 + a1 * c2;

            U k4 = d1 * c2 + c1 * d2;
            
            U k5 = b1 * a2 + a1 * b2;

            sdata[threadIdx.x] += dx * ((a1a2 * x0 * x0) + 
                                  dx * ((x0 * (2.0 * a1a2 + x0 * k5)) / 2.0 +
                                  dx * ((a1a2 + x0 * (2.0 * k5 + k3 * x0)) / 3.0 + 
                                  dx * ((k5 + x0 * (2.0 * k3 + k2 * x0)) / 4.0 +
                                  dx * ((k3 + x0 * (2.0 * k2 + k1 * x0)) / 5.0 + 
                                  dx * ((k2 + x0 * (2.0 * k1 + k4 * x0)) / 6.0 + 
                                  dx * ((k1 + x0 * (2.0 * k4 + d1d2 * x0)) / 7.0 + 
                                  dx * ((k4 + 2.0 * d1d2 * x0) / 8.0 + 
                                  dx * d1d2 / 9.0)))))))); 
        }
    }
    __syncthreads();

    //for (int s = 1; s < blockDim.x; s *= 2) 
    //{
    //    if (threadIdx.x % (2 * s) == 0) sdata[threadIdx.x] += sdata[threadIdx.x + s];
    //    __syncthreads();
    //}
    
    if (threadIdx.x == 0) for (int i = 1; i < blockDim.x; i++) sdata[0] += sdata[i];

    return sdata[0];
}

template <> __device__ 
hipDoubleComplex spline_inner_product_gpu_function<double, hipDoubleComplex>(int ld, int size, double* r_dr, 
                                                                           double* s1_coefs, 
                                                                           hipDoubleComplex* s2_coefs)
{
    int N = size / blockDim.x;
    if (size % blockDim.x != 0) N++;

    extern __shared__ char sdata_ptr[];
    hipDoubleComplex* sdata = (hipDoubleComplex*)&sdata_ptr[0];

    int a_offs = 0 * ld;
    int b_offs = 1 * ld;
    int c_offs = 2 * ld;
    int d_offs = 3 * ld;

    sdata[threadIdx.x] = make_hipDoubleComplex(0.0, 0.0);

    for (int n = 0; n < N; n++)
    {
        int i = n * blockDim.x + threadIdx.x;
        if (i < size - 1)
        {
            double x0 = r_dr[i];
            double dx = r_dr[ld + i];

            double a1 = s1_coefs[a_offs + i];
            double b1 = s1_coefs[b_offs + i];
            double c1 = s1_coefs[c_offs + i];
            double d1 = s1_coefs[d_offs + i];
            
            hipDoubleComplex a2 = s2_coefs[a_offs + i];
            hipDoubleComplex b2 = s2_coefs[b_offs + i];
            hipDoubleComplex c2 = s2_coefs[c_offs + i];
            hipDoubleComplex d2 = s2_coefs[d_offs + i];
                
            hipDoubleComplex a1a2 = make_hipDoubleComplex(a1 * a2.x, a1 * a2.y);
            hipDoubleComplex d1d2 = make_hipDoubleComplex(d1 * d2.x, d1 * d2.y);
                
            hipDoubleComplex k1 = make_hipDoubleComplex(d1 * b2.x + c1 * c2.x + b1 * d2.x, 
                                                      d1 * b2.y + c1 * c2.y + b1 * d2.y);

            hipDoubleComplex k2 = make_hipDoubleComplex(d1 * a2.x + c1 * b2.x + b1 * c2.x + a1 * d2.x, 
                                                      d1 * a2.y + c1 * b2.y + b1 * c2.y + a1 * d2.y);

            hipDoubleComplex k3 = make_hipDoubleComplex(c1 * a2.x + b1 * b2.x + a1 * c2.x, 
                                                      c1 * a2.y + b1 * b2.y + a1 * c2.y);

            hipDoubleComplex k4 = make_hipDoubleComplex(d1 * c2.x + c1 * d2.x, d1 * c2.y + c1 * d2.y);
            
            hipDoubleComplex k5 = make_hipDoubleComplex(b1 * a2.x + a1 * b2.x, b1 * a2.y + a1 * b2.y);

            hipDoubleComplex z = make_hipDoubleComplex(
                                  dx * ((a1a2.x * x0 * x0) + 
                                  dx * ((x0 * (2.0 * a1a2.x + x0 * k5.x)) / 2.0 +
                                  dx * ((a1a2.x + x0 * (2.0 * k5.x + k3.x * x0)) / 3.0 + 
                                  dx * ((k5.x + x0 * (2.0 * k3.x + k2.x * x0)) / 4.0 +
                                  dx * ((k3.x + x0 * (2.0 * k2.x + k1.x * x0)) / 5.0 + 
                                  dx * ((k2.x + x0 * (2.0 * k1.x + k4.x * x0)) / 6.0 + 
                                  dx * ((k1.x + x0 * (2.0 * k4.x + d1d2.x * x0)) / 7.0 + 
                                  dx * ((k4.x + 2.0 * d1d2.x * x0) / 8.0 + 
                                  dx * d1d2.x / 9.0)))))))),
                                  dx * ((a1a2.y * x0 * x0) + 
                                  dx * ((x0 * (2.0 * a1a2.y + x0 * k5.y)) / 2.0 +
                                  dx * ((a1a2.y + x0 * (2.0 * k5.y + k3.y * x0)) / 3.0 + 
                                  dx * ((k5.y + x0 * (2.0 * k3.y + k2.y * x0)) / 4.0 +
                                  dx * ((k3.y + x0 * (2.0 * k2.y + k1.y * x0)) / 5.0 + 
                                  dx * ((k2.y + x0 * (2.0 * k1.y + k4.y * x0)) / 6.0 + 
                                  dx * ((k1.y + x0 * (2.0 * k4.y + d1d2.y * x0)) / 7.0 + 
                                  dx * ((k4.y + 2.0 * d1d2.y * x0) / 8.0 + 
                                  dx * d1d2.y / 9.0)))))))));

            sdata[threadIdx.x] = hipCadd(sdata[threadIdx.x], z);
        }
    }
    __syncthreads();

    ////for (int s = 1; s < blockDim.x; s *= 2) 
    ////{
    ////    if (threadIdx.x % (2 * s) == 0) sdata[threadIdx.x] += sdata[threadIdx.x + s];
    ////    __syncthreads();
    ////}
    //
    if (threadIdx.x == 0) for (int i = 1; i < blockDim.x; i++) sdata[0] = hipCadd(sdata[0], sdata[i]);

    return sdata[0];
}

template <typename T, typename U>
__global__ void spline_inner_product_gpu_kernel(int ld, int size, double* r_dr, T* s1_coefs, U* s2_coefs, U* result)
{
    result[0] = spline_inner_product_gpu_function(ld, size, r_dr, s1_coefs, s2_coefs);
}

template <typename T>
void spline_inner_product_gpu(int size, double* r_dr, T* s1_coefs, T* s2_coefs)
{
    dim3 threadsPerBlock(64);
    dim3 numBlocks(1);

    T* d_result;
    hipMalloc(&d_result, 1 * sizeof(T));
    spline_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16>>>(size, size, r_dr, s1_coefs, s2_coefs, d_result);

    T* h_result = (T*)malloc(1 * sizeof(T));
    hipMemcpy(h_result, d_result, 1 * sizeof(T), hipMemcpyDeviceToHost);

    printf("GPU result : %18.12f \n", h_result[0]);

    hipFree(d_result);
    free(h_result);
    
    //hipDoubleComplex* d_zresult;
    //hipMalloc(&d_zresult, 1 * sizeof(hipDoubleComplex));
    //
    //hipDoubleComplex* zs2;
    //hipMalloc(&zs2, size * 4 * sizeof(hipDoubleComplex));
    //
    //for (int i = 0; i < size * 4; i++) zs2[i] = make_hipDoubleComplex(s2_coefs[i], s2_coefs[i]);

    //spline_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16>>>(size, size, r_dr, s1_coefs, zs2, d_zresult);

    //hipDoubleComplex* h_zresult = (hipDoubleComplex*)malloc(1 * sizeof(hipDoubleComplex));
    //hipMemcpy(h_zresult, d_zresult, 1 * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    //printf("GPU result : %18.12f %18.12f\n", h_zresult[0].x, h_zresult[0].y);

    //hipFree(d_zresult);
    //free(h_zresult);
    //free(zs2);
}

template void spline_inner_product_gpu<double>(int size, double* r_dr, double* s1_coefs, double* s2_coefs);




__global__ void bessel_lo_inner_product_gpu_kernel(int max_num_mt_points, int lmax_pw, int num_atom_types, int num_gkvec, 
                                                   double* jl_coefs, double* lo_coefs, int* l_by_ilo, int* iat_by_ilo, 
                                                   int* nmtp_by_ilo, double* r_dr, double* jlo)
{
    int igk = blockIdx.x;
    int ilo = blockIdx.y;
    int l = l_by_ilo[ilo];
    int iat = iat_by_ilo[ilo];
    int nmtp = nmtp_by_ilo[ilo];

    int ld1 = max_num_mt_points * 4;
    int ld2 = ld1 * (lmax_pw + 1);
    int ld3 = ld2 * num_atom_types;
    double* jl_ptr = &jl_coefs[l * ld1 + iat * ld2 + igk * ld3];
    double* lo_ptr = &lo_coefs[ld1 * ilo];
    double* r_dr_ptr = &r_dr[2 * max_num_mt_points * iat];
    
    jlo[igk + ilo * num_gkvec] = spline_inner_product_gpu_function(max_num_mt_points, nmtp, r_dr_ptr, jl_ptr, lo_ptr);
}


void bessel_lo_inner_product_gpu(int num_gkvec, int num_lo, int max_num_mt_points, int lmax_pw, int num_atom_types, 
                                 double* jl_coefs, double* lo_coefs, int* l_by_ilo, int* iat_by_ilo, int* nmtp_by_ilo, 
                                 double* r_dr, double* jlo)
{
    dim3 threadsPerBlock(64);
    dim3 numBlocks(num_gkvec, num_lo);

    bessel_lo_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16>>>
        (max_num_mt_points, lmax_pw, num_atom_types, num_gkvec, jl_coefs, lo_coefs, l_by_ilo, iat_by_ilo, nmtp_by_ilo,
         r_dr, jlo);
}

__global__ void bessel_vlo_inner_product_gpu_kernel(int max_num_mt_points, int lmax_pw, int lmmax_pw, int num_atom_types, 
                                                    int num_gkvec, double* jl_coefs, hipDoubleComplex* vlo_coefs,
                                                    int* l_by_lm, int* iat_by_ilo, int* nmtp_by_ilo, double* r_dr, 
                                                    hipDoubleComplex* jvlo)

{
    int igk = blockIdx.x;
    int ilo = blockIdx.y;
    int lm = blockIdx.z;
    int l = l_by_lm[lm];
    int iat = iat_by_ilo[ilo];
    int nmtp = nmtp_by_ilo[ilo];
    
    int ld1 = max_num_mt_points * 4;
    int ld2 = ld1 * (lmax_pw + 1);
    int ld3 = ld2 * num_atom_types;
    double* jl_ptr = &jl_coefs[l * ld1 + iat * ld2 + igk * ld3];
    hipDoubleComplex* vlo_ptr = &vlo_coefs[ld1 * lm + ld1 * lmmax_pw * ilo];
    double* r_dr_ptr = &r_dr[2 * max_num_mt_points * iat];
    
    jvlo[lm + igk * lmmax_pw + ilo * lmmax_pw * num_gkvec] = 
        spline_inner_product_gpu_function(max_num_mt_points, nmtp, r_dr_ptr, jl_ptr, vlo_ptr);
}



void bessel_vlo_inner_product_gpu(int num_gkvec, int num_lo, int max_num_mt_points, int lmax_pw, int lmmax_pw, int num_atom_types, 
                                 double* jl_coefs, void* vlo_coefs, int* l_by_lm, int* iat_by_ilo, int* nmtp_by_ilo, 
                                 double* r_dr, void* jvlo)
{
    dim3 threadsPerBlock(128);
    dim3 numBlocks(num_gkvec, num_lo, lmmax_pw);

    bessel_vlo_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 128 * 16>>>
        (max_num_mt_points, lmax_pw, lmmax_pw, num_atom_types, num_gkvec, jl_coefs, (hipDoubleComplex*)vlo_coefs, 
         l_by_lm, iat_by_ilo, nmtp_by_ilo, r_dr, (hipDoubleComplex*)jvlo);
}

__global__ void bessel_vlm_inner_product_gpu_kernel(int max_num_mt_points, int lmax_pot, int lmmax_pot, int* iat_by_ia,
                                                    int* nmtp_by_ia, int* l_by_lm, double* r_dr, double* bessel, 
                                                    double* vlm, double* vjlm)
{
    int lm = blockIdx.x;
    int ia = blockIdx.y;
    int iat = iat_by_ia[ia];
    int nmtp = nmtp_by_ia[ia];
    int l = l_by_lm[lm];

    int ld1 = max_num_mt_points * 4;
    
    double* jl_ptr = &bessel[l * ld1 + iat * ld1 * (lmax_pot + 1)];
    double* vlm_ptr = &vlm[lm * ld1 + ia * ld1 * lmmax_pot];
    double* r_dr_ptr = &r_dr[2 * max_num_mt_points * iat];

    vjlm[lm + ia * lmmax_pot] = spline_inner_product_gpu_function(max_num_mt_points, nmtp, r_dr_ptr, jl_ptr, vlm_ptr);
}


void bessel_vlm_inner_product_gpu(int max_num_mt_points, int lmax_pot, int lmmax_pot, int num_atoms, int num_atom_types, 
                                  int* iat_by_ia, int* nmtp_by_ia, int* l_by_lm, double* r_dr, double* jl_coefs, 
                                  double* vlm_coefs, double* vjlm, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];
    dim3 threadsPerBlock(64);
    dim3 numBlocks(lmmax_pot, num_atoms);
    
    bessel_vlm_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16, stream>>>
        (max_num_mt_points, lmax_pot, lmmax_pot, iat_by_ia, nmtp_by_ia, l_by_lm, r_dr, jl_coefs, vlm_coefs, vjlm);
}









