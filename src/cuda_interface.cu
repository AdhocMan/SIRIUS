#include "hip/hip_runtime.h"
// This file must be compiled with nvcc

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <map>
#include <string>
#include <vector>

const double twopi = 6.2831853071795864769;

class cuda_timers_wrapper
{
    private:

        std::map<std::string, std::vector<float> > cuda_timers_;

    public:

        void add_measurment(const std::string& label, float value)
        {
            cuda_timers_[label].push_back(value);
        }

        void print()
        {
            printf("\n");
            printf("CUDA timers (ms)\n");
            for (int i = 0; i < 115; i++) printf("-");
            printf("\n");
            printf("name                                                              count      total        min        max    average\n");
            for (int i = 0; i < 115; i++) printf("-");
            printf("\n");

            std::map<std::string, std::vector<float> >::iterator it;
            for (it = cuda_timers_.begin(); it != cuda_timers_.end(); it++)
            {
                int count = (int)it->second.size();
                double total = 0.0;
                float minval = 1e10;
                float maxval = 0.0;
                for (int i = 0; i < count; i++)
                {
                    total += it->second[i];
                    minval = std::min(minval, it->second[i]);
                    maxval = std::max(maxval, it->second[i]);
                }
                double average = (count == 0) ? 0.0 : total / count;
                if (count == 0) minval = 0.0;

                printf("%-60s :    %5i %10.4f %10.4f %10.4f %10.4f\n", it->first.c_str(), count, total, minval, maxval, average);
            }
        }
};

cuda_timers_wrapper cuda_timers;

class cuda_timer
{
    private:

        hipEvent_t e_start_;
        hipEvent_t e_stop_;
        bool active_;
        std::string label_;

        void start()
        {
            hipEventCreate(&e_start_);
            hipEventCreate(&e_stop_);
            hipEventRecord(e_start_, 0);
        }

        void stop()
        {
            float time;
            hipEventRecord(e_stop_, 0);
            hipEventSynchronize(e_stop_);
            hipEventElapsedTime(&time, e_start_, e_stop_);
            hipEventDestroy(e_start_);
            hipEventDestroy(e_stop_);
            cuda_timers.add_measurment(label_, time);
            active_ = false;
        }

    public:

        cuda_timer(const std::string& label__) : label_(label__), active_(false)
        {
            start();
        }

        ~cuda_timer()
        {
            stop();
        }
};

extern "C" void print_cuda_timers()
{
    cuda_timers.print();
}

//=====================
// Auxiliary functions
//=====================

__device__ size_t array2D_offset(int i0, int i1, int ld0)
{
    return i0 + i1 * ld0;
}

// TODO: can be optimized in terms of multiplication
__device__ size_t array3D_offset(int i0, int i1, int i2, int ld0, int ld1)
{
    return i0 + i1 * ld0 + i2 * ld0 * ld1;
}

// TODO: can be optimized in terms of multiplication
__device__ size_t array4D_offset(int i0, int i1, int i2, int i3, int ld0, int ld1, int ld2)
{
    return i0 + i1 * ld0 + i2 * ld0 * ld1 + i3 * ld0 * ld1 * ld2;
}

inline __host__ __device__ int num_blocks(int length, int block_size)
{
    return (length / block_size) + min(length % block_size, 1);
}

//================
// CUDA functions
//================

extern "C" void cuda_malloc(void** ptr, size_t size)
{
    if (hipMalloc(ptr, size) != hipSuccess)
    {
        printf("failed to execute hipMalloc() \n");
        exit(0);
    }
}

extern "C" void cuda_free(void* ptr)
{
    if (hipFree(ptr) != hipSuccess)
    {
        printf("failed to execute hipFree() \n");
        exit(0);
    }
}

extern "C" void cuda_malloc_host(void** ptr, size_t size)
{
    if (hipHostMalloc(ptr, size) != hipSuccess)
    {  
        printf("hipHostMalloc failed\n");
        exit(-1);
    }
}

extern "C" void cuda_free_host(void** ptr)
{
    if (hipHostFree(*ptr) != hipSuccess)
    {
        printf("hipHostFree failed\n");
        exit(-1);
    }
}

extern "C" void cuda_copy_to_device(void* target, void* source, size_t size)
{
    if (hipMemcpy(target, source, size, hipMemcpyHostToDevice) != hipSuccess)
    {
        printf("failed to execute hipMemcpy(hipMemcpyHostToDevice)\n");
        exit(0);
    }
}

extern "C" void cuda_copy_to_host(void* target, void* source, size_t size)
{
    if (hipMemcpy(target, source, size, hipMemcpyDeviceToHost) != hipSuccess)
    {
        printf("failed to execute hipMemcpy(hipMemcpyDeviceToHost)\n");
        exit(0);
    }
}

extern "C" void cuda_device_synchronize()
{
    if (hipDeviceSynchronize() != hipSuccess)
    {
        printf("failed to execute hipDeviceSynchronize()\n");
        exit(0);
    }
}

extern "C" void cuda_device_reset()
{
    if (hipDeviceReset() != hipSuccess)
    {
        printf("faile to execute hipDeviceReset()\n");
        exit(0);
    }
}

hipStream_t* streams;

extern "C" void cuda_create_streams(int num_streams)
{
    streams = (hipStream_t*)malloc(num_streams * sizeof(hipStream_t));
    //for (int i = 0; i < num_streams; i++) hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
    for (int i = 0; i < num_streams; i++) hipStreamCreate(&streams[i]);
}

extern "C" void cuda_destroy_streams(int num_streams)
{
    for (int i = 0; i < num_streams; i++) hipStreamDestroy(streams[i]);
    free(streams);
}

extern "C" void cuda_stream_synchronize(int stream_id)
{
    if (hipStreamSynchronize(streams[stream_id]) != hipSuccess)
    {
        printf("failed to execute hipStreamSynchronize()\n");
        exit(0);
    }
}

extern "C" void cuda_async_copy_to_device(void* target, void* source, size_t size, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];

    if (hipMemcpyAsync(target, source, size, hipMemcpyHostToDevice, stream) != hipSuccess)
    {
        printf("failed to execute hipMemcpy(hipMemcpyHostToDevice)\n");
        exit(0);
    }
}

extern "C" void cuda_async_copy_to_host(void* target, void* source, size_t size, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];

    if (hipMemcpyAsync(target, source, size, hipMemcpyDeviceToHost, stream) != hipSuccess)
    {
        printf("failed to execute hipMemcpy(hipMemcpyDeviceToHost)\n");
        exit(0);
    }
}

extern "C" void cuda_memset(void* ptr, int value, size_t size)
{
    if (hipMemset(ptr, value, size) != hipSuccess)
    {
        printf("failed to execute hipMemset()\n");
        exit(0);
    }
}

extern "C" void cuda_host_register(void* ptr, size_t size)
{
    assert(ptr);
    
    hipError_t err = hipHostRegister(ptr, size, 0);
    if (err != hipSuccess)
    {
        printf("failed to execute hipHostRegister\n");
        switch (err)
        {
            case hipErrorInvalidValue:
                printf("hipErrorInvalidValue\n");
                break;
            case hipErrorOutOfMemory:
                printf("hipErrorOutOfMemory\n");
                break;
            default:
                printf("unrecognized error\n");
        }
        exit(-1);
    }
}

extern "C" void cuda_host_unregister(void* ptr)
{
    if (hipHostUnregister(ptr) != hipSuccess)
    {
        printf("failed to execute hipHostUnregister\n");
        exit(-1);
    }
}

//* hipDeviceProp_t& cuda_devprop()
//* {
//*     static hipDeviceProp_t devprop;
//* 
//*     return devprop;
//* }

extern "C" size_t cuda_get_free_mem()
{
    size_t free, total;
    
    hipMemGetInfo(&free, &total);

    return free;
}

extern "C" void cuda_device_info()
{
    int count;
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("failed to execute hipGetDeviceCount() \n");
        exit(-1);
    }

    if (count == 0)
    {
        printf("no avaiable devices\n");
        exit(-1);
    }

    hipDeviceProp_t devprop;
     
    if (hipGetDeviceProperties(&devprop, 0) != hipSuccess)
    {
        printf("failed to execute hipGetDeviceProperties()\n");
        exit(-1);
    }
    
    printf("name                        : %s \n", devprop.name);
    printf("major                       : %i \n", devprop.major);
    printf("minor                       : %i \n", devprop.minor);
    printf("asyncEngineCount            : %i \n", devprop.asyncEngineCount);
    printf("canMapHostMemory            : %i \n", devprop.canMapHostMemory);
    printf("clockRate                   : %i kHz \n", devprop.clockRate);
    printf("concurrentKernels           : %i \n", devprop.concurrentKernels);
    printf("ECCEnabled                  : %i \n", devprop.ECCEnabled);
    printf("l2CacheSize                 : %i kB \n", devprop.l2CacheSize/1024);
    printf("maxGridSize                 : %i %i %i \n", devprop.maxGridSize[0], devprop.maxGridSize[1], devprop.maxGridSize[2]);
    printf("maxThreadsDim               : %i %i %i \n", devprop.maxThreadsDim[0], devprop.maxThreadsDim[1], devprop.maxThreadsDim[2]);
    printf("maxThreadsPerBlock          : %i \n", devprop.maxThreadsPerBlock);
    printf("maxThreadsPerMultiProcessor : %i \n", devprop.maxThreadsPerMultiProcessor);
    printf("memoryBusWidth              : %i bits \n", devprop.memoryBusWidth);
    printf("memoryClockRate             : %i kHz \n", devprop.memoryClockRate);
    printf("memPitch                    : %zi \n", devprop.memPitch);
    printf("multiProcessorCount         : %i \n", devprop.multiProcessorCount);
    printf("regsPerBlock                : %i \n", devprop.regsPerBlock);
    printf("sharedMemPerBlock           : %li kB \n", devprop.sharedMemPerBlock/1024);
    printf("totalConstMem               : %li kB \n", devprop.totalConstMem/1024);
    printf("totalGlobalMem              : %li kB \n", devprop.totalGlobalMem/1024);
    printf("available memory            : %li kB \n", cuda_get_free_mem() / 1024);
}

//==================
// CUBLAS functions
//==================

hipblasHandle_t& cublas_handle()
{
    static hipblasHandle_t handle;
    static bool init = false;

    if (!init)
    {
        if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
        {
            printf("hipblasCreate() failed \n");
            exit(-1);
        }
        init = true;
    }
    
    return handle;
}

extern "C" void cublas_init()
{
    cublas_handle();
}

extern "C" void cublas_zgemm(int transa, int transb, int32_t m, int32_t n, int32_t k, 
                             void* alpha, void* a, int32_t lda, void* b, 
                             int32_t ldb, void* beta, void* c, int32_t ldc)
{
    const hipblasOperation_t trans[] = {HIPBLAS_OP_N, HIPBLAS_OP_T, HIPBLAS_OP_C};
    
    
    hipblasStatus_t status = hipblasZgemm(cublas_handle(), trans[transa], trans[transb], m, n, k, (hipDoubleComplex*)alpha, 
                                        (hipDoubleComplex*)a, lda, (hipDoubleComplex*)b, ldb, (hipDoubleComplex*)beta, 
                                        (hipDoubleComplex*)c, ldc);
    if (status == HIPBLAS_STATUS_SUCCESS) return;

    printf("failed to execute hipblasZgemm\n");
    
    switch (status)
    {
        case HIPBLAS_STATUS_NOT_INITIALIZED:
        {
            printf("the library was not initialized\n");
            break;
        }
        case HIPBLAS_STATUS_INVALID_VALUE:
        {
            printf("the parameters m,n,k<0\n");
            break;
        }
        case HIPBLAS_STATUS_ARCH_MISMATCH:
        {
            printf("he device does not support double-precision\n");
            break;
        }
        case HIPBLAS_STATUS_EXECUTION_FAILED:
        {
            printf("the function failed to launch on the GPU\n");
            break;
        }
    }

    exit(-1);
}

// A(GPU) => B(CPU)
extern "C" void cublas_get_matrix(int rows, int cols, int elemSize, const void *A_device, int lda, void *B_host, int ldb)
{
    if (hipblasGetMatrix(rows, cols, elemSize, A_device, lda, B_host, ldb) != HIPBLAS_STATUS_SUCCESS)
    {
        printf("failed to execute hipblasGetMatrix\n");
        exit(-1);
    }
}

extern "C" void cublas_get_matrix_async(int rows, int cols, int elemSize, const void *A_device, int lda, void *B_host, int ldb, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];

    if (hipblasGetMatrixAsync(rows, cols, elemSize, A_device, lda, B_host, ldb, stream) != HIPBLAS_STATUS_SUCCESS)
    {
        printf("failed to execute hipblasGetMatrix\n");
        exit(-1);
    }
}

// A(CPU) => B(GPU)
extern "C" void cublas_set_matrix(int rows, int cols, int elemSize, const void *A_host, int lda, void *B_device, int ldb)
{
    if (hipblasSetMatrix(rows, cols, elemSize, A_host, lda, B_device, ldb) != HIPBLAS_STATUS_SUCCESS)
    {
        printf("failed to execute hipblasSetMatrix\n");
        exit(-1);
    }
}

extern "C" void cublas_set_matrix_async(int rows, int cols, int elemSize, const void *A_host, int lda, void *B_device, int ldb, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];

    if (hipblasSetMatrixAsync(rows, cols, elemSize, A_host, lda, B_device, ldb, stream) != HIPBLAS_STATUS_SUCCESS)
    {
        printf("failed to execute hipblasSetMatrix\n");
        exit(-1);
    }
}

//=================
// CUFFT functions
//=================

hipfftHandle plan;
int nfft_of_plan;
int size_of_plan;
hipDoubleComplex* fft_buffer = NULL;

extern "C" void cufft_create_batch_plan(int nx, int ny, int nz, int nfft, void* fft_buffer__)
{
    int fft_size = nx * ny * nz;
    int n[] = {nz, ny, nx};

    hipfftResult result = hipfftPlanMany(&plan, 3, n, n, 1, fft_size, n, 1, fft_size, HIPFFT_Z2Z, nfft);
    if (result != HIPFFT_SUCCESS)
    {
        printf("failed to execute hipfftPlanMany()\n");
        exit(0);
    }

    nfft_of_plan = nfft;
    size_of_plan = fft_size;

    fft_buffer = (hipDoubleComplex*)fft_buffer__;
}

extern "C" void cufft_destroy_batch_plan()
{
    hipfftDestroy(plan);
}

//== __global__ void cufft_batch_load_kernel(int fft_size, int num_gkvec, int* map, hipDoubleComplex* phi, 
//==                                         hipDoubleComplex* fft_buffer)
//== {
//==     int i = blockIdx.y;
//==     int ig = blockDim.x * blockIdx.x + threadIdx.x;
//== 
//==     if (ig < num_gkvec) fft_buffer[array2D_offset(map[ig], i, fft_size)] = phi[array2D_offset(ig, i, num_gkvec)];
//== }

//= __global__ void cufft_batch_apply_v_kernel(int fft_size, hipDoubleComplex* v_r, hipDoubleComplex* fft_buffer)
//= {
//=     int i = blockIdx.y;
//=     int ir = blockDim.x * blockIdx.x + threadIdx.x;
//=     if (ir < fft_size) 
//=     {
//=         fft_buffer[array2D_offset(ir, i, fft_size)] = 
//=             hipCmul(fft_buffer[array2D_offset(ir, i, fft_size)], v_r[ir]);
//=     }
//= }

//== __global__ void cufft_batch_unload_kernel(int fft_size, int num_gkvec, int* map, hipDoubleComplex* fft_buffer,
//==                                           hipDoubleComplex* phi)
//== {
//==     int i = blockIdx.y;
//==     int ig = blockDim.x * blockIdx.x + threadIdx.x;
//== 
//==     if (ig < num_gkvec) 
//==     {
//==         phi[array2D_offset(ig, i, num_gkvec)] = 
//==             hipCdiv(fft_buffer[array2D_offset(map[ig], i, fft_size)], make_hipDoubleComplex(double(fft_size), 0));
//==     }
//== }

//== extern "C" void cufft_batch_apply_v(int fft_size, int num_gkvec, int num_phi, void* buffer, int* map, void* v_r, void* p)
//== {
//==     dim3 threadsPerBlock(64);
//==     dim3 numBlocks(num_blocks(num_gkvec, 64), num_phi);
//==     
//==     cuda_memset(buffer, 0, fft_size * num_phi * sizeof(hipDoubleComplex));
//== 
//==     cufft_batch_load_kernel<<<numBlocks, threadsPerBlock>>>
//==         (fft_size, num_gkvec, map, (hipDoubleComplex*)p, (hipDoubleComplex*)buffer);
//==     
//==     hipfftExecZ2Z(plan, (hipfftDoubleComplex*)buffer, (hipfftDoubleComplex*)buffer, HIPFFT_BACKWARD);
//==     
//==     //dim3 numBlocks_r(num_blocks(fft_size, 64), num_phi);
//==     //cufft_batch_apply_v_kernel<<<numBlocks_r, threadsPerBlock>>>
//==     //    (fft_size, (hipDoubleComplex*)v_r, (hipDoubleComplex*)buffer);
//==     
//==     hipfftExecZ2Z(plan, (hipfftDoubleComplex*)buffer, (hipfftDoubleComplex*)buffer, HIPFFT_FORWARD);
//== 
//==     cufft_batch_unload_kernel<<<numBlocks, threadsPerBlock>>>
//==         (fft_size, num_gkvec, map, (hipDoubleComplex*)buffer, (hipDoubleComplex*)p);
//== }

__global__ void cufft_batch_load_gpu_kernel(int fft_size, 
                                            int num_elements, 
                                            int* map, 
                                            hipDoubleComplex* data, 
                                            hipDoubleComplex* fft_buffer)
{
    int i = blockIdx.y;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elements) fft_buffer[array2D_offset(map[idx], i, fft_size)] = data[array2D_offset(idx, i, num_elements)];
}

extern "C" void cufft_batch_load_gpu(int num_elements, int* map, void* data)
{
    dim3 threadsPerBlock(64);
    dim3 numBlocks(num_blocks(num_elements, 64), nfft_of_plan);
    
    cuda_memset(fft_buffer, 0, size_of_plan * nfft_of_plan * sizeof(hipDoubleComplex));

    cufft_batch_load_gpu_kernel<<<numBlocks, threadsPerBlock>>>(size_of_plan, 
                                                                num_elements, 
                                                                map, 
                                                                (hipDoubleComplex*)data, 
                                                                fft_buffer);
}

__global__ void cufft_batch_unload_gpu_kernel(int fft_size, 
                                              int num_elements, 
                                              int* map, 
                                              hipDoubleComplex* data, 
                                              hipDoubleComplex* fft_buffer)
{
    int i = blockIdx.y;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elements) 
    {
        data[array2D_offset(idx, i, num_elements)] = 
            hipCdiv(fft_buffer[array2D_offset(map[idx], i, fft_size)], make_hipDoubleComplex(double(fft_size), 0));
    }
}

extern "C" void cufft_batch_unload_gpu(int num_elements, int* map, void* data)
{
    dim3 threadsPerBlock(64);
    dim3 numBlocks(num_blocks(num_elements, 64), nfft_of_plan);
    
    cufft_batch_unload_gpu_kernel<<<numBlocks, threadsPerBlock>>>(size_of_plan, 
                                                                  num_elements, 
                                                                  map, 
                                                                  (hipDoubleComplex*)data, 
                                                                  fft_buffer);
}

__global__ void cufft_normalize(int size, hipDoubleComplex* buffer)
{
    int i = blockIdx.y;
    int ir = blockDim.x * blockIdx.x + threadIdx.x;

    if (ir < size) 
    {
        buffer[array2D_offset(ir, i, size)] = 
            hipCdiv(buffer[array2D_offset(ir, i, size)], make_hipDoubleComplex(double(size), 0));
    }
}

extern "C" void cufft_forward_transform()
{
    hipfftExecZ2Z(plan, fft_buffer, fft_buffer, HIPFFT_FORWARD);
    
    //== dim3 threadsPerBlock(64);
    //== dim3 numBlocks(num_blocks(size_of_plan, 64), nfft_of_plan);
    //== cufft_normalize<<<numBlocks, threadsPerBlock>>>(size_of_plan, (hipfftDoubleComplex*)buffer);
}

extern "C" void cufft_backward_transform()
{
    hipfftExecZ2Z(plan, fft_buffer, fft_buffer, HIPFFT_BACKWARD);
}

//==================================
// High-level functions and kernels
//==================================

template <typename T, typename U>
__device__ U spline_inner_product_gpu_function(int ld, int size, double* r_dr, T* s1_coefs, U* s2_coefs)
{
    int N = size / blockDim.x;
    if (size % blockDim.x != 0) N++;

    extern __shared__ char sdata_ptr[];
    U* sdata = (U*)&sdata_ptr[0];

    int a_offs = 0 * ld;
    int b_offs = 1 * ld;
    int c_offs = 2 * ld;
    int d_offs = 3 * ld;

    sdata[threadIdx.x] = 0;

    for (int n = 0; n < N; n++)
    {
        int i = n * blockDim.x + threadIdx.x;
        if (i < size - 1)
        {
            double x0 = r_dr[i];
            double dx = r_dr[ld + i];

            T a1 = s1_coefs[a_offs + i];
            T b1 = s1_coefs[b_offs + i];
            T c1 = s1_coefs[c_offs + i];
            T d1 = s1_coefs[d_offs + i];
            
            U a2 = s2_coefs[a_offs + i];
            U b2 = s2_coefs[b_offs + i];
            U c2 = s2_coefs[c_offs + i];
            U d2 = s2_coefs[d_offs + i];
                
            U a1a2 = a1 * a2;
            U d1d2 = d1 * d2;
                
            U k1 = d1 * b2 + c1 * c2 + b1 * d2;

            U k2 = d1 * a2 + c1 * b2 + b1 * c2 + a1 * d2;

            U k3 = c1 * a2 + b1 * b2 + a1 * c2;

            U k4 = d1 * c2 + c1 * d2;
            
            U k5 = b1 * a2 + a1 * b2;

            sdata[threadIdx.x] += dx * ((a1a2 * x0 * x0) + 
                                  dx * ((x0 * (2.0 * a1a2 + x0 * k5)) / 2.0 +
                                  dx * ((a1a2 + x0 * (2.0 * k5 + k3 * x0)) / 3.0 + 
                                  dx * ((k5 + x0 * (2.0 * k3 + k2 * x0)) / 4.0 +
                                  dx * ((k3 + x0 * (2.0 * k2 + k1 * x0)) / 5.0 + 
                                  dx * ((k2 + x0 * (2.0 * k1 + k4 * x0)) / 6.0 + 
                                  dx * ((k1 + x0 * (2.0 * k4 + d1d2 * x0)) / 7.0 + 
                                  dx * ((k4 + 2.0 * d1d2 * x0) / 8.0 + 
                                  dx * d1d2 / 9.0)))))))); 
        }
    }
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) 
    {
        if (threadIdx.x % (2 * s) == 0) sdata[threadIdx.x] += sdata[threadIdx.x + s];
        __syncthreads();
    }
    
    //if (threadIdx.x == 0) for (int i = 1; i < blockDim.x; i++) sdata[0] += sdata[i];

    return sdata[0];
}

template <> __device__ 
hipDoubleComplex spline_inner_product_gpu_function<double, hipDoubleComplex>(int ld, int size, double* r_dr, 
                                                                           double* s1_coefs, 
                                                                           hipDoubleComplex* s2_coefs)
{
    int N = size / blockDim.x;
    if (size % blockDim.x != 0) N++;

    extern __shared__ char sdata_ptr[];
    hipDoubleComplex* sdata = (hipDoubleComplex*)&sdata_ptr[0];

    int a_offs = 0 * ld;
    int b_offs = 1 * ld;
    int c_offs = 2 * ld;
    int d_offs = 3 * ld;

    sdata[threadIdx.x] = make_hipDoubleComplex(0.0, 0.0);

    for (int n = 0; n < N; n++)
    {
        int i = n * blockDim.x + threadIdx.x;
        if (i < size - 1)
        {
            double x0 = r_dr[i];
            double dx = r_dr[ld + i];

            double a1 = s1_coefs[a_offs + i];
            double b1 = s1_coefs[b_offs + i];
            double c1 = s1_coefs[c_offs + i];
            double d1 = s1_coefs[d_offs + i];
            
            hipDoubleComplex a2 = s2_coefs[a_offs + i];
            hipDoubleComplex b2 = s2_coefs[b_offs + i];
            hipDoubleComplex c2 = s2_coefs[c_offs + i];
            hipDoubleComplex d2 = s2_coefs[d_offs + i];
                
            hipDoubleComplex a1a2 = make_hipDoubleComplex(a1 * a2.x, a1 * a2.y);
            hipDoubleComplex d1d2 = make_hipDoubleComplex(d1 * d2.x, d1 * d2.y);
                
            hipDoubleComplex k1 = make_hipDoubleComplex(d1 * b2.x + c1 * c2.x + b1 * d2.x, 
                                                      d1 * b2.y + c1 * c2.y + b1 * d2.y);

            hipDoubleComplex k2 = make_hipDoubleComplex(d1 * a2.x + c1 * b2.x + b1 * c2.x + a1 * d2.x, 
                                                      d1 * a2.y + c1 * b2.y + b1 * c2.y + a1 * d2.y);

            hipDoubleComplex k3 = make_hipDoubleComplex(c1 * a2.x + b1 * b2.x + a1 * c2.x, 
                                                      c1 * a2.y + b1 * b2.y + a1 * c2.y);

            hipDoubleComplex k4 = make_hipDoubleComplex(d1 * c2.x + c1 * d2.x, d1 * c2.y + c1 * d2.y);
            
            hipDoubleComplex k5 = make_hipDoubleComplex(b1 * a2.x + a1 * b2.x, b1 * a2.y + a1 * b2.y);

            hipDoubleComplex z = make_hipDoubleComplex(
                                  dx * ((a1a2.x * x0 * x0) + 
                                  dx * ((x0 * (2.0 * a1a2.x + x0 * k5.x)) / 2.0 +
                                  dx * ((a1a2.x + x0 * (2.0 * k5.x + k3.x * x0)) / 3.0 + 
                                  dx * ((k5.x + x0 * (2.0 * k3.x + k2.x * x0)) / 4.0 +
                                  dx * ((k3.x + x0 * (2.0 * k2.x + k1.x * x0)) / 5.0 + 
                                  dx * ((k2.x + x0 * (2.0 * k1.x + k4.x * x0)) / 6.0 + 
                                  dx * ((k1.x + x0 * (2.0 * k4.x + d1d2.x * x0)) / 7.0 + 
                                  dx * ((k4.x + 2.0 * d1d2.x * x0) / 8.0 + 
                                  dx * d1d2.x / 9.0)))))))),
                                  dx * ((a1a2.y * x0 * x0) + 
                                  dx * ((x0 * (2.0 * a1a2.y + x0 * k5.y)) / 2.0 +
                                  dx * ((a1a2.y + x0 * (2.0 * k5.y + k3.y * x0)) / 3.0 + 
                                  dx * ((k5.y + x0 * (2.0 * k3.y + k2.y * x0)) / 4.0 +
                                  dx * ((k3.y + x0 * (2.0 * k2.y + k1.y * x0)) / 5.0 + 
                                  dx * ((k2.y + x0 * (2.0 * k1.y + k4.y * x0)) / 6.0 + 
                                  dx * ((k1.y + x0 * (2.0 * k4.y + d1d2.y * x0)) / 7.0 + 
                                  dx * ((k4.y + 2.0 * d1d2.y * x0) / 8.0 + 
                                  dx * d1d2.y / 9.0)))))))));

            sdata[threadIdx.x] = hipCadd(sdata[threadIdx.x], z);
        }
    }
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) 
    {
        if (threadIdx.x % (2 * s) == 0) sdata[threadIdx.x] = hipCadd(sdata[threadIdx.x], sdata[threadIdx.x + s]);
        __syncthreads();
    }
    
    //if (threadIdx.x == 0) for (int i = 1; i < blockDim.x; i++) sdata[0] = hipCadd(sdata[0], sdata[i]);

    return sdata[0];
}

template <typename T, typename U>
__global__ void spline_inner_product_gpu_kernel(int ld, int size, double* r_dr, T* s1_coefs, U* s2_coefs, U* result)
{
    result[0] = spline_inner_product_gpu_function(ld, size, r_dr, s1_coefs, s2_coefs);
}

template <typename T>
void spline_inner_product_gpu(int size, double* r_dr, T* s1_coefs, T* s2_coefs)
{
    dim3 threadsPerBlock(64);
    dim3 numBlocks(1);

    T* d_result;
    hipMalloc(&d_result, 1 * sizeof(T));
    spline_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16>>>(size, size, r_dr, s1_coefs, s2_coefs, d_result);

    T* h_result = (T*)malloc(1 * sizeof(T));
    hipMemcpy(h_result, d_result, 1 * sizeof(T), hipMemcpyDeviceToHost);

    printf("GPU result : %18.12f \n", h_result[0]);

    hipFree(d_result);
    free(h_result);
    
    //hipDoubleComplex* d_zresult;
    //hipMalloc(&d_zresult, 1 * sizeof(hipDoubleComplex));
    //
    //hipDoubleComplex* zs2;
    //hipMalloc(&zs2, size * 4 * sizeof(hipDoubleComplex));
    //
    //for (int i = 0; i < size * 4; i++) zs2[i] = make_hipDoubleComplex(s2_coefs[i], s2_coefs[i]);

    //spline_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16>>>(size, size, r_dr, s1_coefs, zs2, d_zresult);

    //hipDoubleComplex* h_zresult = (hipDoubleComplex*)malloc(1 * sizeof(hipDoubleComplex));
    //hipMemcpy(h_zresult, d_zresult, 1 * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    //printf("GPU result : %18.12f %18.12f\n", h_zresult[0].x, h_zresult[0].y);

    //hipFree(d_zresult);
    //free(h_zresult);
    //free(zs2);
}

template void spline_inner_product_gpu<double>(int size, double* r_dr, double* s1_coefs, double* s2_coefs);







// Input array dimensions:
//   sbessel_coefs(max_num_mt_points * 4, lmax_pw + 1, num_atom_types, num_gkvec_row);
//   lo_coefs(max_num_mt_points * 4, num_lo);
//   jlo(num_gkvec, num_lo);
__global__ void sbessel_lo_inner_product_gpu_kernel(int* kargs, int num_gkvec, int* l_by_ilo, int* iat_by_ilo, 
                                                    int* nmtp_by_iat, double* r_dr, double* sbessel_coefs, 
                                                    double* lo_coefs, double* jlo)
{
    int num_atom_types = kargs[0];
    int max_nmtp = kargs[1];
    int lmax_pw = kargs[2];

    int igk = blockIdx.x;
    int ilo = blockIdx.y;

    int l = l_by_ilo[ilo];
    int iat = iat_by_ilo[ilo];
    int nmtp = nmtp_by_iat[iat];

    double* jl_ptr = &sbessel_coefs[array4D_offset(0, l, iat, igk, max_nmtp * 4, lmax_pw + 1, num_atom_types)];
    double* lo_ptr = &lo_coefs[array2D_offset(0, ilo, max_nmtp * 4)];
    double* r_dr_ptr = &r_dr[array2D_offset(0, iat, 2 * max_nmtp)];
    
    jlo[array2D_offset(igk, ilo, num_gkvec)] = 
        spline_inner_product_gpu_function(max_nmtp, nmtp, r_dr_ptr, jl_ptr, lo_ptr);
}


void sbessel_lo_inner_product_gpu(int* kargs, int num_gkvec, int num_lo, int* l_by_ilo, int* iat_by_ilo, 
                                  int* nmtp_by_iat, double* r_dr, double* sbessel_coefs, double* lo_coefs, double* jlo)
{
    dim3 threadsPerBlock(64);
    dim3 numBlocks(num_gkvec, num_lo);

    sbessel_lo_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16>>>
        (kargs, num_gkvec, l_by_ilo, iat_by_ilo, nmtp_by_iat, r_dr, sbessel_coefs, lo_coefs, jlo);
}

// Compute <jl|V|lo>
// Input array dimensions:
//   vlo(max_num_mt_points * 4, lmmax_pw, num_lo_col)
//   jvlo(lmmax_pw, num_gkvec, num_lo)
__global__ void sbessel_vlo_inner_product_gpu_kernel(int* kargs, int num_gkvec, int* l_by_lm, int* iat_by_ilo, 
                                                     int* nmtp_by_iat, double* r_dr, double* sbessel_coefs, 
                                                     hipDoubleComplex* vlo_coefs, hipDoubleComplex* jvlo)
{
    int num_atom_types = kargs[0];
    int max_nmtp = kargs[1];
    int lmax_pw = kargs[2];
    int lmmax_pw = kargs[3];

    int igk = blockIdx.x;
    int ilo = blockIdx.y;
    int lm = blockIdx.z;

    int l = l_by_lm[lm];
    int iat = iat_by_ilo[ilo];
    int nmtp = nmtp_by_iat[iat];
    
    double* jl_ptr = &sbessel_coefs[array4D_offset(0, l, iat, igk, max_nmtp * 4, lmax_pw + 1, num_atom_types)];
    hipDoubleComplex* vlo_ptr = &vlo_coefs[array3D_offset(0, lm, ilo, 4 * max_nmtp, lmmax_pw)];
    double* r_dr_ptr = &r_dr[array2D_offset(0, iat, 2 * max_nmtp)];
    
    jvlo[array3D_offset(lm, igk, ilo, lmmax_pw, num_gkvec)] = 
        spline_inner_product_gpu_function(max_nmtp, nmtp, r_dr_ptr, jl_ptr, vlo_ptr);
}

// Compute <jl|V|lo>
void sbessel_vlo_inner_product_gpu(int* kargs, int num_gkvec, int num_lo, int lmmax_pw, int* l_by_lm, int* iat_by_ilo, 
                                   int* nmtp_by_iat, double* r_dr, double* sbessel_coefs, void* vlo_coefs, void* jvlo)
{
    dim3 threadsPerBlock(64);
    dim3 numBlocks(num_gkvec, num_lo, lmmax_pw);

    sbessel_vlo_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16>>>
        (kargs, num_gkvec, l_by_lm, iat_by_ilo, nmtp_by_iat, r_dr, sbessel_coefs, (hipDoubleComplex*)vlo_coefs, 
         (hipDoubleComplex*)jvlo);
}

__global__ void sbessel_vlm_inner_product_gpu_kernel(int* kargs, int* iat_by_ia, int* l_by_lm, int* nmtp_by_iat,
                                                     double* r_dr, double* sbessel_coefs, double* vlm_coefs, 
                                                     double* jvlm)
{
    int max_nmtp = kargs[1];
    int lmax_pot = kargs[2];
    int lmmax_pot = kargs[3];
    
    int lm = blockIdx.x;
    int ia = blockIdx.y;

    int iat = iat_by_ia[ia];
    int nmtp = nmtp_by_iat[ia];
    int l = l_by_lm[lm];

    double* jl_ptr = &sbessel_coefs[array3D_offset(0, l, iat, max_nmtp * 4, lmax_pot + 1)];
    double* vlm_ptr = &vlm_coefs[array3D_offset(0, lm, ia, max_nmtp * 4, lmmax_pot)];
    double* r_dr_ptr = &r_dr[array2D_offset(0, iat, 2 * max_nmtp)];

    jvlm[array2D_offset(lm, ia, lmmax_pot)] = 
        spline_inner_product_gpu_function(max_nmtp, nmtp, r_dr_ptr, jl_ptr, vlm_ptr);
}


void sbessel_vlm_inner_product_gpu(int* kargs, int lmmax_pot, int num_atoms, int* iat_by_ia, int* l_by_lm, 
                                   int* nmtp_by_iat, double* r_dr, double* sbessel_coefs, double* vlm_coefs, 
                                   double* jvlm, int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];
    dim3 threadsPerBlock(64);
    dim3 numBlocks(lmmax_pot, num_atoms);
    
    sbessel_vlm_inner_product_gpu_kernel<<<numBlocks, threadsPerBlock, 64 * 16, stream>>>
        (kargs, iat_by_ia, l_by_lm, nmtp_by_iat, r_dr, sbessel_coefs, vlm_coefs, jvlm);
}


//__global__ void add_band_density_gpu_kernel(int nmtp, int lmmax_rho, int max_nmtp, int max_num_gaunt, int* gaunt12_size, 
//                                            int* gaunt12_lm1_by_lm3, int* gaunt12_lm2_by_lm3, 
//                                            hipDoubleComplex* gaunt12_cg, hipDoubleComplex* fylm, double weight, 
//                                            int ia, double* dens)
//{
//    int ir = blockDim.x * blockIdx.x + threadIdx.x;
//    int lm = blockIdx.y;
//
//    int offs3 = array3D_offset(ir, lm, ia, max_nmtp, lmmax_rho);
//
//    if (ir < nmtp)
//    {
//        for (int k = 0; k < gaunt12_size[lm]; k++)
//        {
//            int offs = array2D_offset(k, lm, max_num_gaunt);
//            int lm1 = gaunt12_lm1_by_lm3[offs];
//            int lm2 = gaunt12_lm2_by_lm3[offs];
//            hipDoubleComplex cg = gaunt12_cg[offs];
//            
//            int offs1 = array2D_offset(ir, lm1, max_nmtp);
//            int offs2 = array2D_offset(ir, lm2, max_nmtp);
//
//            hipDoubleComplex z = hipCmul(hipConj(fylm[offs1]), fylm[offs2]);
//
//            dens[offs3] += weight * hipCreal(hipCmul(z, cg));
//        }
//    }
//}

__global__ void add_band_density_gpu_kernel(int lmmax_rho, int lmmax_wf, int max_nmtp, int* ia_by_ialoc, 
                                            int* iat_by_ia, int* nmtp_by_iat, int max_num_gaunt, 
                                            int* gaunt12_size, int* gaunt12_lm1_by_lm3, int* gaunt12_lm2_by_lm3, 
                                            hipDoubleComplex* gaunt12_cg, hipDoubleComplex* fylm, double weight, 
                                            double* dens)
{
    int lm = blockIdx.x;
    int ialoc = blockIdx.y;
    int ia = ia_by_ialoc[ialoc];
    int iat = iat_by_ia[ia];
    int nmtp = nmtp_by_iat[iat];

    int offs3 = array3D_offset(0, lm, ialoc, max_nmtp, lmmax_rho);

    int N = nmtp / blockDim.x;
    if (nmtp % blockDim.x != 0) N++;

    for (int k = 0; k < gaunt12_size[lm]; k++)
    {
        int offs = array2D_offset(k, lm, max_num_gaunt);

        int lm1 = gaunt12_lm1_by_lm3[offs];
        int lm2 = gaunt12_lm2_by_lm3[offs];
        hipDoubleComplex cg = gaunt12_cg[offs];
        
        int offs1 = array3D_offset(0, lm1, ia, max_nmtp, lmmax_wf);
        int offs2 = array3D_offset(0, lm2, ia, max_nmtp, lmmax_wf);
        
        for (int n = 0; n < N; n++)
        {
            int ir = n * blockDim.x + threadIdx.x;
            if (ir < nmtp)
            {
                hipDoubleComplex z = hipCmul(hipConj(fylm[offs1 + ir]), fylm[offs2 + ir]);

                dens[offs3 + ir] += weight * hipCreal(hipCmul(z, cg));
            }
        }
    }
}

void add_band_density_gpu(int lmmax_rho, int lmmax_wf, int max_nmtp, int num_atoms_loc, int* ia_by_ialoc, 
                          int* iat_by_ia, int* nmtp_by_iat, int max_num_gaunt, int* gaunt12_size, 
                          int* gaunt12_lm1_by_lm3, int* gaunt12_lm2_by_lm3, void* gaunt12_cg, void* fylm, 
                          double weight, double* dens)
{
    dim3 threadsPerBlock(128);
    dim3 numBlocks(lmmax_rho, num_atoms_loc);
    add_band_density_gpu_kernel<<<numBlocks, threadsPerBlock>>>
        (lmmax_rho, lmmax_wf, max_nmtp, ia_by_ialoc, iat_by_ia, nmtp_by_iat, max_num_gaunt, gaunt12_size, 
         gaunt12_lm1_by_lm3, gaunt12_lm2_by_lm3, (hipDoubleComplex*)gaunt12_cg, (hipDoubleComplex*)fylm, weight, dens);
}
    


__global__ void scale_matrix_columns_gpu_kernel(int nrow, hipDoubleComplex* mtrx, double* a)
{
    int icol = blockIdx.y;
    int irow = blockIdx.x * blockDim.x + threadIdx.x;
    if (irow < nrow) 
    {
        mtrx[array2D_offset(irow, icol, nrow)] =
            hipCmul(mtrx[array2D_offset(irow, icol, nrow)], make_hipDoubleComplex(a[icol], 0));
    }
}

// scale each column of the matrix by a column-dependent constant
extern "C" void scale_matrix_columns_gpu(int nrow, int ncol, void* mtrx, double* a)
{
    dim3 threadsPerBlock(64);
    dim3 numBlocks(num_blocks(nrow, 64), ncol);
    scale_matrix_columns_gpu_kernel<<<numBlocks, threadsPerBlock>>>(nrow, (hipDoubleComplex*)mtrx, a);
}

__global__ void scale_matrix_rows_gpu_kernel(int nrow, hipDoubleComplex* mtrx, double* v)
{
    int icol = blockIdx.y;
    int irow = blockDim.x * blockIdx.x + threadIdx.x;
    if (irow < nrow) 
    {
        mtrx[array2D_offset(irow, icol, nrow)] = 
            hipCmul(mtrx[array2D_offset(irow, icol, nrow)], make_hipDoubleComplex(v[irow], 0));
    }
}

// scale each row of the matrix by a row-dependent constant
extern "C" void scale_matrix_rows_gpu(int nrow, int ncol, void* mtrx, double* v)
{
    dim3 threadsPerBlock(64);
    dim3 numBlocks(num_blocks(nrow, 64), ncol);

    scale_matrix_rows_gpu_kernel<<<
        numBlocks, 
        threadsPerBlock>>>(nrow, 
                           (hipDoubleComplex*)mtrx, 
                           v);
}

__global__ void create_beta_pw_gpu_kernel(int num_gkvec, 
                                          int* beta_t_idx, 
                                          hipDoubleComplex* beta_pw_type, 
                                          double* gkvec, 
                                          double* atom_pos,
                                          hipDoubleComplex* beta_pw)
{
    int i = blockIdx.y;
    int ia = beta_t_idx[array2D_offset(0, i, 2)];
    int offset_t = beta_t_idx[array2D_offset(1, i, 2)];

    int igk = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (igk < num_gkvec)
    {
        double p = 0;
        for (int x = 0; x < 3; x++) p += atom_pos[array2D_offset(x, ia, 3)] * gkvec[array2D_offset(x, igk, 3)];
        p *= twopi;
        
        double sinp = sin(p);
        double cosp = cos(p);

        beta_pw[array2D_offset(igk, i, num_gkvec)] = 
            hipCmul(beta_pw_type[array2D_offset(igk, offset_t, num_gkvec)], make_hipDoubleComplex(cosp, -sinp));
    }
}

extern "C" void create_beta_pw_gpu(int num_gkvec, 
                                   int num_beta_atot, 
                                   int* beta_t_idx,
                                   void* beta_pw_type,
                                   double* gkvec,
                                   double* atom_pos,
                                   void* beta_pw)
{
    dim3 threadsPerBlock(64);
    dim3 numBlocks(num_blocks(num_gkvec, 64), num_beta_atot);

    create_beta_pw_gpu_kernel<<<
        numBlocks, 
        threadsPerBlock>>>(num_gkvec, 
                           beta_t_idx, 
                           (hipDoubleComplex*)beta_pw_type,
                           gkvec,
                           atom_pos,
                           (hipDoubleComplex*)beta_pw);
}

//== __global__ void create_beta_pw_gpu_kernel(int num_gkvec, 
//==                                           int beta_a_ofs,
//==                                           int* beta_t_idx, 
//==                                           hipDoubleComplex* beta_pw_type, 
//==                                           double* gkvec, 
//==                                           double* atom_pos,
//==                                           hipDoubleComplex* beta_pw)
//== {
//== 
//==     int i = blockIdx.y;
//==     int ia = beta_t_idx[array2D_offset(0, i + beta_a_ofs, 2)];
//==     int offset_t = beta_t_idx[array2D_offset(1, i + beta_a_ofs, 2)];
//== 
//==     int igk = blockDim.x * blockIdx.x + threadIdx.x;
//==     
//==     if (igk < num_gkvec)
//==     {
//==         double p = 0;
//==         for (int x = 0; x < 3; x++) p += atom_pos[array2D_offset(x, ia, 3)] * gkvec[array2D_offset(igk, x, num_gkvec)];
//==         p *= twopi;
//==         
//==         double sinp = sin(p);
//==         double cosp = cos(p);
//== 
//==         beta_pw[array2D_offset(igk, i, num_gkvec)] = 
//==             hipCmul(beta_pw_type[array2D_offset(igk, offset_t, num_gkvec)], make_hipDoubleComplex(cosp, -sinp));
//==     }
//== }
//== 
//== extern "C" void create_single_beta_pw_gpu(int num_gkvec, 
//==                                           int num_beta_a, 
//==                                           int beta_a_ofs, 
//==                                           int* beta_t_idx,
//==                                           void* beta_pw_type,
//==                                           double* gkvec,
//==                                           double* atom_pos,
//==                                           void* beta_pw)
//== {
//==     dim3 threadsPerBlock(64);
//==     dim3 numBlocks(num_blocks(num_gkvec, 64), num_beta_a);
//== 
//==     create_beta_pw_gpu_kernel<<<numBlocks, threadsPerBlock>>>(num_gkvec,
//==                                                               beta_a_ofs,
//==                                                               beta_t_idx, 
//==                                                               (hipDoubleComplex*)beta_pw_type,
//==                                                               gkvec,
//==                                                               atom_pos,
//==                                                               (hipDoubleComplex*)beta_pw);
//== }

//== #define BLOCK_SIZE 32
//== 
//== __global__ void generate_beta_phi_gpu_kernel(int num_gkvec, 
//==                                              int num_beta,
//==                                              int num_phi,
//==                                              int* beta_t_idx, 
//==                                              double* atom_pos, 
//==                                              double* gkvec, 
//==                                              hipDoubleComplex* beta_pw_type,
//==                                              hipDoubleComplex* phi,
//==                                              hipDoubleComplex* beta_phi)
//== {
//==     int idx_beta = blockDim.x * blockIdx.x + threadIdx.x;
//==     int idx_phi = blockDim.y * blockIdx.y + threadIdx.y;
//==     int ia, offset_t;
//==     double x0, y0, z0;
//== 
//==     if (idx_beta < num_beta)
//==     {
//==         ia = beta_t_idx[array2D_offset(0, idx_beta, 2)];
//==         offset_t = beta_t_idx[array2D_offset(1, idx_beta, 2)];
//==         x0 = atom_pos[array2D_offset(0, ia, 3)];
//==         y0 = atom_pos[array2D_offset(1, ia, 3)];
//==         z0 = atom_pos[array2D_offset(2, ia, 3)];
//==     }
//== 
//==     int N = num_blocks(num_gkvec, BLOCK_SIZE);
//== 
//==     hipDoubleComplex val = make_hipDoubleComplex(0.0, 0.0);
//== 
//==     for (int m = 0; m < N; m++)
//==     {
//==         __shared__ hipDoubleComplex beta_pw_tile[BLOCK_SIZE][BLOCK_SIZE];
//==         __shared__ hipDoubleComplex phi_tile[BLOCK_SIZE][BLOCK_SIZE];
//== 
//==         int bs = (m + 1) * BLOCK_SIZE > num_gkvec ? num_gkvec - m * BLOCK_SIZE : BLOCK_SIZE;
//== 
//==         int igk = m * BLOCK_SIZE + threadIdx.y;
//== 
//==         if (igk < num_gkvec && idx_beta < num_beta)
//==         {
//==             double x1 = gkvec[array2D_offset(igk, 0, num_gkvec)];
//==             double y1 = gkvec[array2D_offset(igk, 1, num_gkvec)];
//==             double z1 = gkvec[array2D_offset(igk, 2, num_gkvec)];
//== 
//==             double p = twopi * (x0 * x1 + y0 * y1 + z0 * z1);
//==             double sinp = sin(p);
//==             double cosp = cos(p);
//== 
//==             beta_pw_tile[threadIdx.x][threadIdx.y] = hipCmul(hipConj(beta_pw_type[array2D_offset(igk, offset_t, num_gkvec)]), 
//==                                                             make_hipDoubleComplex(cosp, sinp));
//== 
//==         }
//==         
//==         igk = m * BLOCK_SIZE + threadIdx.x;
//== 
//==         if (igk < num_gkvec && idx_phi < num_phi)
//==             phi_tile[threadIdx.y][threadIdx.x] = phi[array2D_offset(igk, idx_phi, num_gkvec)];
//== 
//==         __syncthreads();
//== 
//==         for (int i = 0; i < bs; i++) val = hipCadd(val, hipCmul(beta_pw_tile[threadIdx.x][i], phi_tile[threadIdx.y][i]));
//== 
//==         __syncthreads();
//==     }
//== 
//==     if (idx_beta < num_beta && idx_phi < num_phi) beta_phi[array2D_offset(idx_beta, idx_phi, num_beta)] = val;
//== }
//== 
//== 
//== extern "C" void generate_beta_phi_gpu(int num_gkvec, 
//==                                       int num_beta, 
//==                                       int num_phi, 
//==                                       int* beta_t_idx, 
//==                                       double* atom_pos,
//==                                       double* gkvec,
//==                                       void* beta_pw_type,
//==                                       void* phi,
//==                                       void* beta_phi)
//== {
//== 
//==     dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
//==     dim3 numBlocks(num_blocks(num_beta, BLOCK_SIZE), num_blocks(num_phi, BLOCK_SIZE));
//== 
//==     generate_beta_phi_gpu_kernel<<<
//==         numBlocks, 
//==         threadsPerBlock>>>(num_gkvec, 
//==                            num_beta,
//==                            num_phi,
//==                            beta_t_idx, 
//==                            atom_pos,
//==                            gkvec, 
//==                            (hipDoubleComplex*)beta_pw_type,
//==                            (hipDoubleComplex*)phi,
//==                            (hipDoubleComplex*)beta_phi);
//== }

__global__ void restore_valence_density_gpu_kernel(int num_gvec_loc,
                                                   int* atom_type,
                                                   int* num_beta, 
                                                   double* atom_pos,
                                                   int* gvec,
                                                   hipDoubleComplex* pp_complex_density_matrix,
                                                   int ldm,
                                                   hipDoubleComplex** q_pw,
                                                   hipDoubleComplex* f_pw)
{
    extern __shared__ char sdata_ptr[];
    hipDoubleComplex* sdata = (hipDoubleComplex*)&sdata_ptr[0];

    int ia = blockIdx.x;

    int iat = atom_type[ia];

    int nbf = num_beta[iat];

    hipDoubleComplex* q_pw_t = q_pw[iat];
    //printf("ia : %i, type : %i, nbf : %i, q_pw : %p", ia, iat, nbf, q_pw_t);

    double ax = atom_pos[array2D_offset(0, ia, 3)];
    double ay = atom_pos[array2D_offset(1, ia, 3)];
    double az = atom_pos[array2D_offset(2, ia, 3)];

    if (threadIdx.x == 0)
    {
        for (int xi2 = 0; xi2 < nbf; xi2++)
        {
            for (int xi1 = 0; xi1 <= xi2; xi1++)
            {
                int idx12 = xi2 * (xi2 + 1) / 2 + xi1;
                sdata[idx12] = pp_complex_density_matrix[array4D_offset(xi2, xi1, 0, ia, ldm, ldm, 1)];
            }
        }
    }
    __syncthreads();

    hipDoubleComplex* f_pw_a = &f_pw[array2D_offset(0, ia, num_gvec_loc)];
    
    int N = num_blocks(num_gvec_loc, blockDim.x);

    for (int n = 0; n < N; n++)
    {
        int igloc = n * blockDim.x + threadIdx.x;
        if (igloc < num_gvec_loc)
        {
            int gvx = gvec[array2D_offset(0, igloc, 3)];
            int gvy = gvec[array2D_offset(1, igloc, 3)];
            int gvz = gvec[array2D_offset(2, igloc, 3)];

            double p = twopi * (ax * gvx + ay * gvy + az * gvz);
            
            double sinp = sin(p);
            double cosp = cos(p);

            hipDoubleComplex zval = make_hipDoubleComplex(0.0, 0.0);

            // \sum_{xi1, xi2} D_{xi2,xi1} * Q(G)_{xi1, xi2}
            for (int xi2 = 0; xi2 < nbf; xi2++)
            {
                int idx12 = xi2 * (xi2 + 1) / 2;

                //hipDoubleComplex q = hipCmul(make_hipDoubleComplex(cosp, -sinp), q_pw_t[array2D_offset(igloc, idx12 + xi2, num_gvec_loc)]);

                // add diagonal term
                //f_pw_a[igloc] = hipCadd(f_pw_a[igloc], hipCmul(sdata[idx12 + xi2], q));
                zval = hipCadd(zval, hipCmul(sdata[idx12 + xi2], q_pw_t[array2D_offset(igloc, idx12 + xi2, num_gvec_loc)]));

                // add non-diagonal terms
                for (int xi1 = 0; xi1 < xi2; xi1++, idx12++)
                {
                    hipDoubleComplex q = q_pw_t[array2D_offset(igloc, idx12, num_gvec_loc)];
                    //q = hipCmul(make_hipDoubleComplex(cosp, -sinp), q_pw_t[array2D_offset(igloc, idx12, num_gvec_loc)]);
                    
                    //double d = 2 * hipCreal(hipCmul(sdata[idx12], q));

                    //f_pw_a[igloc] = hipCadd(f_pw_a[igloc], make_hipDoubleComplex(d, 0));
                    //double d = 2 * hipCreal(hipCmul(sdata[idx12], q_pw_t[array2D_offset(igloc, idx12, num_gvec_loc)])
                    zval.x += 2 * (sdata[idx12].x * q.x - sdata[idx12].y * q.y);
                    //zval = hipCadd(zval, make_hipDoubleComplex(2 * hipCreal(hipCmul(sdata[idx12], q_pw_t[array2D_offset(igloc, idx12, num_gvec_loc)])), 0.0));
                }
            }
            f_pw_a[igloc] = hipCadd(f_pw_a[igloc], hipCmul(zval, make_hipDoubleComplex(cosp, -sinp))); 
        }
    }
}

__global__ void reduce_rho_pw_kernel(int num_atoms, int num_gvec_loc, hipDoubleComplex* f_pw, hipDoubleComplex* rho_pw)
{
    int igloc = blockDim.x * blockIdx.x + threadIdx.x;

    if (igloc < num_gvec_loc)
    {
        for (int ia = 0; ia < num_atoms; ia++) 
            rho_pw[igloc] = hipCadd(rho_pw[igloc], f_pw[array2D_offset(igloc, ia, num_gvec_loc)]);
    }
}


extern "C" void restore_valence_density_gpu(int num_atoms, 
                                            int num_gvec_loc,
                                            int* atom_type,
                                            int* num_beta, 
                                            double* atom_pos, 
                                            int* gvec,
                                            void* pp_complex_density_matrix,
                                            int ldm,
                                            void** q_pw,
                                            void* rho_pw)
{
    dim3 threadsPerBlock(1024);
    dim3 numBlocks(num_atoms);

    hipDoubleComplex* f_pw;
    cuda_malloc((void**)&f_pw, num_gvec_loc * num_atoms * sizeof(hipDoubleComplex));
    cuda_memset(f_pw, 0, num_gvec_loc * num_atoms * sizeof(hipDoubleComplex));

    restore_valence_density_gpu_kernel<<<
        numBlocks,
        threadsPerBlock,
        sizeof(hipDoubleComplex) * ldm * (ldm + 1) / 2>>>(num_gvec_loc,
                                                         atom_type,
                                                         num_beta, 
                                                         atom_pos, 
                                                         gvec, 
                                                         (hipDoubleComplex*)pp_complex_density_matrix,
                                                         ldm,
                                                         (hipDoubleComplex**)q_pw,
                                                         f_pw);
    
    cuda_memset(rho_pw, 0, num_gvec_loc * sizeof(hipDoubleComplex));
    
    dim3 grid_t(128);
    dim3 grid_b(num_blocks(num_gvec_loc, grid_t.x));
    reduce_rho_pw_kernel<<<grid_b, grid_t>>>
        (num_atoms, num_gvec_loc, f_pw, (hipDoubleComplex*)rho_pw);
    
    cuda_device_synchronize();
    cuda_free(f_pw);
}




__global__ void restore_valence_density_gpu_kernel_v2(int num_gvec_loc,
                                                      int num_beta, 
                                                      double ax,
                                                      double ay,
                                                      double az,
                                                      int* gvec,
                                                      hipDoubleComplex* pp_complex_density_matrix,
                                                      int ldm,
                                                      hipDoubleComplex* q_pw_t,
                                                      hipDoubleComplex* rho_pw)
{
    extern __shared__ char sdata_ptr[];
    hipDoubleComplex* sdata = (hipDoubleComplex*)&sdata_ptr[0];

    if (threadIdx.x == 0)
    {
        for (int xi2 = 0; xi2 < num_beta; xi2++)
        {
            for (int xi1 = 0; xi1 <= xi2; xi1++)
            {
                int idx12 = xi2 * (xi2 + 1) / 2 + xi1;
                sdata[idx12] = pp_complex_density_matrix[array3D_offset(xi2, xi1, 0, ldm, ldm)];
            }
        }
    }
    __syncthreads();

    int igloc = blockIdx.x * blockDim.x + threadIdx.x;
    if (igloc < num_gvec_loc)
    {
        int gvx = gvec[array2D_offset(0, igloc, 3)];
        int gvy = gvec[array2D_offset(1, igloc, 3)];
        int gvz = gvec[array2D_offset(2, igloc, 3)];

        double p = twopi * (ax * gvx + ay * gvy + az * gvz);
        
        double sinp = sin(p);
        double cosp = cos(p);

        hipDoubleComplex zval = make_hipDoubleComplex(0.0, 0.0);

        // \sum_{xi1, xi2} D_{xi2,xi1} * Q(G)_{xi1, xi2}
        for (int xi2 = 0; xi2 < num_beta; xi2++)
        {
            int idx12 = xi2 * (xi2 + 1) / 2;

            // add diagonal term
            zval = hipCadd(zval, hipCmul(sdata[idx12 + xi2], q_pw_t[array2D_offset(igloc, idx12 + xi2, num_gvec_loc)]));

            // add non-diagonal terms
            for (int xi1 = 0; xi1 < xi2; xi1++, idx12++)
            {
                hipDoubleComplex q = q_pw_t[array2D_offset(igloc, idx12, num_gvec_loc)];
                zval.x += 2 * (sdata[idx12].x * q.x - sdata[idx12].y * q.y);
            }
        }
        rho_pw[igloc] = hipCadd(rho_pw[igloc], hipCmul(zval, make_hipDoubleComplex(cosp, -sinp))); 
    }
}

extern "C" void restore_valence_density_gpu_v2(int num_gvec_loc,
                                               int num_beta,
                                               double ax,
                                               double ay,
                                               double az,
                                               int* gvec,
                                               void* pp_complex_density_matrix,
                                               int ldm,
                                               void* q_pw_t,
                                               void* rho_pw,
                                               int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec_loc, grid_t.x));

    restore_valence_density_gpu_kernel_v2<<<grid_b, grid_t, sizeof(hipDoubleComplex) * ldm * (ldm + 1) / 2, stream>>>
        (num_gvec_loc, num_beta, ax, ay, az, gvec, (hipDoubleComplex*)pp_complex_density_matrix, ldm,
         (hipDoubleComplex*)q_pw_t, (hipDoubleComplex*)rho_pw);
}

__global__ void mul_veff_with_phase_factors_kernel(int num_gvec_loc,
                                                   hipDoubleComplex* veff, 
                                                   int* gvec, 
                                                   double ax, 
                                                   double ay, 
                                                   double az, 
                                                   hipDoubleComplex* vtmp)
{
    int igloc = blockDim.x * blockIdx.x + threadIdx.x;
    if (igloc < num_gvec_loc)
    {
        int gvx = gvec[array2D_offset(0, igloc, 3)];
        int gvy = gvec[array2D_offset(1, igloc, 3)];
        int gvz = gvec[array2D_offset(2, igloc, 3)];

        double p = twopi * (ax * gvx + ay * gvy + az * gvz);
            
        vtmp[igloc] = hipCmul(veff[igloc], make_hipDoubleComplex(cos(p), sin(p)));
    }
}
 
extern "C" void mul_veff_with_phase_factors(int num_gvec_loc, 
                                            void* veff, 
                                            int* gvec, 
                                            double ax,
                                            double ay,
                                            double az,
                                            void* vtmp)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec_loc, grid_t.x));

    mul_veff_with_phase_factors_kernel<<<grid_b, grid_t>>>
        (num_gvec_loc, (hipDoubleComplex*)veff, gvec, ax, ay, az, (hipDoubleComplex*)vtmp);
}

__global__ void compute_d_mtrx_gpu_kernel(int num_gvec_loc, 
                                          hipDoubleComplex* vtmp, 
                                          hipDoubleComplex* q_pw, 
                                          hipDoubleComplex* d_mtrx_gpu)
{
    int idx = blockIdx.x;

    int N = num_blocks(num_gvec_loc, blockDim.x);

    extern __shared__ char sdata_ptr[];
    hipDoubleComplex* sdata = (hipDoubleComplex*)&sdata_ptr[0];

    sdata[threadIdx.x] = make_hipDoubleComplex(0.0, 0.0);

    for (int n = 0; n < N; n++)
    {
        int igloc = n * blockDim.x + threadIdx.x;
        if (igloc < num_gvec_loc)
        {
            sdata[threadIdx.x] = hipCadd(sdata[threadIdx.x], 
                                        hipCmul(vtmp[igloc], 
                                               hipConj(q_pw[array2D_offset(igloc, idx,  num_gvec_loc)])));
        }
    }
    
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) 
    {
        if (threadIdx.x % (2 * s) == 0) sdata[threadIdx.x] = hipCadd(sdata[threadIdx.x], sdata[threadIdx.x + s]);
        __syncthreads();
    }

    d_mtrx_gpu[idx] = sdata[0];
}

extern "C" void compute_d_mtrx_gpu(int num_gvec_loc,
                                   int num_elements,
                                   void* vtmp,
                                   void* q_pw, 
                                   void* d_mtrx_gpu)
{
    dim3 grid_t(64);
    dim3 grid_b(num_elements);

    compute_d_mtrx_gpu_kernel<<<grid_b, grid_t, grid_t.x * sizeof(hipDoubleComplex)>>>
        (num_gvec_loc, (hipDoubleComplex*)vtmp, (hipDoubleComplex*)q_pw, (hipDoubleComplex*)d_mtrx_gpu);
}


extern "C" void compute_d_mtrx_valence_gpu(int num_gvec_loc,
                                           int num_elements,
                                           void* veff, 
                                           int* gvec, 
                                           double ax,
                                           double ay,
                                           double az,
                                           void* vtmp,
                                           void* q_pw_t,
                                           void* d_mtrx,
                                           int stream_id)
{
    hipStream_t stream = (stream_id == -1) ? NULL : streams[stream_id];

    dim3 grid_t(64);

    dim3 grid_b(num_blocks(num_gvec_loc, grid_t.x));
    mul_veff_with_phase_factors_kernel<<<grid_b, grid_t, 0, stream>>>
        (num_gvec_loc, (hipDoubleComplex*)veff, gvec, ax, ay, az, (hipDoubleComplex*)vtmp);

    grid_b = dim3(num_elements);
    compute_d_mtrx_gpu_kernel<<<grid_b, grid_t, grid_t.x * sizeof(hipDoubleComplex), stream>>>
        (num_gvec_loc, (hipDoubleComplex*)vtmp, (hipDoubleComplex*)q_pw_t, (hipDoubleComplex*)d_mtrx);
}

__global__ void add_to_d_mtrx_pw_gpu_kernel(int num_gvec_loc, 
                                       int num_beta,
                                       double ax, 
                                       double ay, 
                                       double az, 
                                       int* gvec,
                                       hipDoubleComplex* d_mtrx_packed,
                                       hipDoubleComplex* d_mtrx_pw)
{
    int idx12 = blockIdx.y;
    int igloc = blockIdx.x * blockDim.x + threadIdx.x;

    if (igloc < num_gvec_loc)
    {
        int gvx = gvec[array2D_offset(0, igloc, 3)];
        int gvy = gvec[array2D_offset(1, igloc, 3)];
        int gvz = gvec[array2D_offset(2, igloc, 3)];

        double p = twopi * (ax * gvx + ay * gvy + az * gvz);

        double sinp = sin(p);
        double cosp = cos(p);
        
        d_mtrx_pw[array2D_offset(igloc, idx12, num_gvec_loc)] = 
            hipCadd(d_mtrx_pw[array2D_offset(igloc, idx12, num_gvec_loc)],   
                   hipCmul(d_mtrx_packed[idx12], make_hipDoubleComplex(cosp, -sinp)));
    }
}

extern "C" void add_to_d_mtrx_pw_gpu(int num_gvec_loc,
                                     int num_beta,
                                     double ax, 
                                     double ay,
                                     double az,
                                     int* gvec,
                                     void* d_mtrx_packed,
                                     void* d_mtrx_pw)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec_loc, grid_t.x), num_beta * (num_beta + 1) / 2);

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
 
    hipEventRecord(start, 0);
 
    add_to_d_mtrx_pw_gpu_kernel<<<grid_b, grid_t>>>
        (num_gvec_loc, num_beta, ax, ay, az, gvec, (hipDoubleComplex*)d_mtrx_packed, (hipDoubleComplex*)d_mtrx_pw);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf ("Time for add_to_d_mtrx_pw_gpu_kernel: %f ms\n", time); 
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

//== __global__ void generate_d_mtrx_pw_gpu_kernel(int num_atoms, 
//==                                               int num_gvec_loc, 
//==                                               const double* atom_pos, 
//==                                               const int* gvec,
//==                                               const hipDoubleComplex* d_mtrx_packed,
//==                                               hipDoubleComplex* d_mtrx_pw)
//== {
//==     int idx12 = blockIdx.y;
//==     int igloc = blockIdx.x * blockDim.x + threadIdx.x;
//== 
//==     if (igloc < num_gvec_loc)
//==     {
//==         int gvx = gvec[array2D_offset(0, igloc, 3)];
//==         int gvy = gvec[array2D_offset(1, igloc, 3)];
//==         int gvz = gvec[array2D_offset(2, igloc, 3)];
//==     
//==         hipDoubleComplex zval = make_hipDoubleComplex(0.0, 0.0);
//==         for (int ia = 0; ia < num_atoms; ia++)
//==         {
//==             double ax = atom_pos[array2D_offset(ia, 0, num_atoms)];
//==             double ay = atom_pos[array2D_offset(ia, 1, num_atoms)];
//==             double az = atom_pos[array2D_offset(ia, 2, num_atoms)];
//== 
//==             double p = twopi * (ax * gvx + ay * gvy + az * gvz);
//== 
//==             double sinp = sin(p);
//==             double cosp = cos(p);
//== 
//==             zval = hipCadd(zval, hipCmul(d_mtrx_packed[array2D_offset(ia, idx12, num_atoms)], 
//==                                        make_hipDoubleComplex(cosp, -sinp)));
//== 
//==         }
//==         
//==         d_mtrx_pw[array2D_offset(igloc, idx12, num_gvec_loc)] = zval;
//==     }
//== }

__global__ void generate_d_mtrx_pw_gpu_kernel(int num_atoms, 
                                              int num_gvec_loc, 
                                              int num_gvec_in_block,
                                              const double* atom_pos, 
                                              const int* gvec,
                                              const hipDoubleComplex* d_mtrx_packed,
                                              hipDoubleComplex* d_mtrx_pw)
{
    extern __shared__ char sdata_ptr[];
    double* sdata = (double*)&sdata_ptr[0];
    
    int idx12 = blockIdx.y;

    if (threadIdx.x == 0)
    {
        for (int ia = 0; ia < num_atoms; ia++)
        {
            double ax = atom_pos[array2D_offset(ia, 0, num_atoms)];
            double ay = atom_pos[array2D_offset(ia, 1, num_atoms)];
            double az = atom_pos[array2D_offset(ia, 2, num_atoms)];

            sdata[ia * 5 + 0] = ax;
            sdata[ia * 5 + 1] = ay;
            sdata[ia * 5 + 2] = az;
            sdata[ia * 5 + 3] = hipCreal(d_mtrx_packed[array2D_offset(ia, idx12, num_atoms)]);
            sdata[ia * 5 + 4] = hipCimag(d_mtrx_packed[array2D_offset(ia, idx12, num_atoms)]);
        }
    }
    __syncthreads();

    int N = num_blocks(num_gvec_in_block, blockDim.x);
    
    for (int n = 0; n < N; n++)
    {
        int igloc = blockIdx.x * num_gvec_in_block + n * blockDim.x + threadIdx.x;

        if (igloc < num_gvec_loc)
        {
            int gvx = gvec[array2D_offset(0, igloc, 3)];
            int gvy = gvec[array2D_offset(1, igloc, 3)];
            int gvz = gvec[array2D_offset(2, igloc, 3)];
        
            hipDoubleComplex zval = make_hipDoubleComplex(0.0, 0.0);
            for (int ia = 0; ia < num_atoms; ia++)
            {
                double ax = sdata[ia * 5 + 0]; 
                double ay = sdata[ia * 5 + 1]; 
                double az = sdata[ia * 5 + 2];

                double p = twopi * (ax * gvx + ay * gvy + az * gvz);

                double sinp = sin(p);
                double cosp = cos(p);

                //zval = hipCadd(zval, hipCmul(d_mtrx_packed[array2D_offset(ia, idx12, num_atoms)], 
                //                           make_hipDoubleComplex(cosp, -sinp)));
                zval = hipCadd(zval, hipCmul(make_hipDoubleComplex(sdata[ia * 5 + 3], sdata[ia * 5 + 4]),
                                           make_hipDoubleComplex(cosp, -sinp)));

            }
            
            d_mtrx_pw[array2D_offset(igloc, idx12, num_gvec_loc)] = zval;
        }
    }
}

__global__ void generate_phase_factors_gpu_kernel(int num_gvec_loc, int num_atoms, double* atom_pos, int* gvec, hipDoubleComplex* phase_factors)
{
    int ia = blockIdx.y;
    int igloc = blockIdx.x * blockDim.x + threadIdx.x;

    if (igloc < num_gvec_loc)
    {
        int gvx = gvec[array2D_offset(0, igloc, 3)];
        int gvy = gvec[array2D_offset(1, igloc, 3)];
        int gvz = gvec[array2D_offset(2, igloc, 3)];
    
        double ax = atom_pos[array2D_offset(ia, 0, num_atoms)];
        double ay = atom_pos[array2D_offset(ia, 1, num_atoms)];
        double az = atom_pos[array2D_offset(ia, 2, num_atoms)];

        double p = twopi * (ax * gvx + ay * gvy + az * gvz);

        double sinp = sin(p);
        double cosp = cos(p);

        phase_factors[array2D_offset(igloc, ia, num_gvec_loc)] = make_hipDoubleComplex(cosp, -sinp);
    }
}


extern "C" void generate_d_mtrx_pw_gpu(int num_atoms,
                                       int num_gvec_loc,
                                       int num_beta,
                                       double* atom_pos,
                                       int* gvec,
                                       void* d_mtrx_packed,
                                       void* d_mtrx_pw)
{
    cuda_timer t("generate_d_mtrx_pw_gpu");

    //== dim3 grid_t(32);
    //== //dim3 grid_b(num_blocks(num_gvec_loc, grid_t.x), num_beta * (num_beta + 1) / 2);
    //== dim3 grid_b(4, num_beta * (num_beta + 1) / 2);

    //== generate_d_mtrx_pw_gpu_kernel<<<grid_b, grid_t, 5 * num_atoms * sizeof(double)>>>
    //==     (num_atoms, num_gvec_loc, num_blocks(num_gvec_loc, 4), atom_pos, gvec, (hipDoubleComplex*)d_mtrx_packed, (hipDoubleComplex*)d_mtrx_pw);

    hipDoubleComplex* phase_factors;
    cuda_malloc((void**)&phase_factors, num_gvec_loc * num_atoms * sizeof (hipDoubleComplex));

    dim3 grid_t(32);
    dim3 grid_b(num_blocks(num_gvec_loc, grid_t.x), num_atoms);

    generate_phase_factors_gpu_kernel<<<grid_b, grid_t>>>
        (num_gvec_loc, num_atoms, atom_pos, gvec, phase_factors);
    
    hipDoubleComplex zone = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex zzero = make_hipDoubleComplex(0.0, 0.0);

    cublas_zgemm(0, 0, num_gvec_loc, num_beta * (num_beta + 1) / 2, num_atoms, (void*)&zone, 
                 (void*)phase_factors, num_gvec_loc, (void*)d_mtrx_packed, num_atoms, (void*)&zzero,
                 d_mtrx_pw, num_gvec_loc);

    cuda_free(phase_factors);
}

__global__ void sum_q_pw_d_mtrx_pw_gpu_kernel(int num_gvec_loc,
                                              int num_beta,
                                              hipDoubleComplex* q_pw_t,
                                              hipDoubleComplex* d_mtrx_pw,
                                              hipDoubleComplex* rho_pw)
{
    int igloc = blockIdx.x * blockDim.x + threadIdx.x;
    if (igloc < num_gvec_loc)
    {
        hipDoubleComplex zval = make_hipDoubleComplex(0.0, 0.0);

        // \sum_{xi1, xi2} D_{xi2,xi1} * Q(G)_{xi1, xi2}
        for (int xi2 = 0; xi2 < num_beta; xi2++)
        {
            int idx12 = xi2 * (xi2 + 1) / 2;

            // add diagonal term
            zval = hipCadd(zval, hipCmul(d_mtrx_pw[array2D_offset(igloc, idx12 + xi2, num_gvec_loc)], 
                                       q_pw_t[array2D_offset(igloc, idx12 + xi2, num_gvec_loc)]));

            // add non-diagonal terms
            for (int xi1 = 0; xi1 < xi2; xi1++, idx12++)
            {
                hipDoubleComplex q = q_pw_t[array2D_offset(igloc, idx12, num_gvec_loc)];
                hipDoubleComplex d = d_mtrx_pw[array2D_offset(igloc, idx12, num_gvec_loc)];
                zval.x += 2 * (d.x * q.x - d.y * q.y);
            }
        }
        rho_pw[igloc] = hipCadd(rho_pw[igloc], zval);
    }
}

extern "C" void sum_q_pw_d_mtrx_pw_gpu(int num_gvec_loc,
                                       int num_beta,
                                       void* q_pw_t,
                                       void* d_mtrx_pw,
                                       void* rho_pw)
{
    cuda_timer t("sum_q_pw_d_mtrx_pw_gpu");

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec_loc, grid_t.x));
    
    sum_q_pw_d_mtrx_pw_gpu_kernel<<<grid_b, grid_t>>>
        (num_gvec_loc, num_beta, (hipDoubleComplex*)q_pw_t, (hipDoubleComplex*)d_mtrx_pw, (hipDoubleComplex*)rho_pw);
}
