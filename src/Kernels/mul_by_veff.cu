#include "hip/hip_runtime.h"
// Copyright (c) 2013-2018 Anton Kozhevnikov, Thomas Schulthess
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
//    and the following disclaimer in the documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/** \file mul_by_veff.cu
 *
 *  \brief CUDA kernel to multiply wave-functions by effective potential.
 */

#include "../SDDK/GPU/cuda_common.hpp"
#include "../SDDK/GPU/acc.hpp"
#include <hip/hip_complex.h>

__global__ void mul_by_veff0_gpu_kernel(int                    size__,
                                        double* const*         veff__,
                                        hipDoubleComplex*       buf__)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size__) {
        hipDoubleComplex z = buf__[i];
        double v0 = veff__[0][i];
        buf__[i] = make_hipDoubleComplex(z.x * v0, z.y * v0);
    }
}

__global__ void mul_by_veff1_gpu_kernel(int                    size__,
                                        double* const*         veff__,
                                        hipDoubleComplex*       buf__)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size__) {
        hipDoubleComplex z = buf__[i];
        double v1 = veff__[1][i];
        buf__[i] = make_hipDoubleComplex(z.x * v1, z.y * v1);
    }
}

__global__ void mul_by_veff2_gpu_kernel(int                    size__,
                                        double* const*         veff__,
                                        hipDoubleComplex*       buf__)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size__) {
        hipDoubleComplex z = buf__[i];
        hipDoubleComplex v = make_hipDoubleComplex(veff__[2][i], -veff__[3][i]);  
        buf__[i] = hipCmul(z, v);
    }
}

__global__ void mul_by_veff3_gpu_kernel(int                    size__,
                                        double* const*         veff__,
                                        hipDoubleComplex*       buf__)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size__) {
        hipDoubleComplex z = buf__[i];
        hipDoubleComplex v = make_hipDoubleComplex(veff__[2][i], veff__[3][i]);
        buf__[i] = hipCmul(z, v);
    }
}

extern "C" void mul_by_veff_gpu(int ispn__, int size__, double* const* veff__, hipDoubleComplex* buf__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(size__, grid_t.x));

    switch (ispn__) {
        case 0: {
            mul_by_veff0_gpu_kernel<<<grid_b, grid_t>>>(size__, veff__, buf__);
            break;
        }
        case 1: {
            mul_by_veff1_gpu_kernel<<<grid_b, grid_t>>>(size__, veff__, buf__);
            break;
        }
        case 2: {
            mul_by_veff2_gpu_kernel<<<grid_b, grid_t>>>(size__, veff__, buf__);
            break;
        }

        case 3: {
            mul_by_veff3_gpu_kernel<<<grid_b, grid_t>>>(size__, veff__, buf__);
            break;
        }
    }
}
