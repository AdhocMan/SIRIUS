#include "hip/hip_runtime.h"
#include "kernels_common.h"

__global__ void mul_veff_with_phase_factors_gpu_kernel(int num_gvec_loc__,
                                                       hipDoubleComplex const* veff__, 
                                                       int const* gvec__, 
                                                       double const* atom_pos__, 
                                                       hipDoubleComplex* veff_a__)
{
    int igloc = blockDim.x * blockIdx.x + threadIdx.x;
    int ia = blockIdx.y;

    if (igloc < num_gvec_loc__)
    {
        int gvx = gvec__[array2D_offset(0, igloc, 3)];
        int gvy = gvec__[array2D_offset(1, igloc, 3)];
        int gvz = gvec__[array2D_offset(2, igloc, 3)];
        double ax = atom_pos__[array2D_offset(0, ia, 3)];
        double ay = atom_pos__[array2D_offset(1, ia, 3)];
        double az = atom_pos__[array2D_offset(2, ia, 3)];

        double p = twopi * (ax * gvx + ay * gvy + az * gvz);
            
        veff_a__[array2D_offset(igloc, ia, num_gvec_loc__)] = hipCmul(veff__[igloc], make_hipDoubleComplex(cos(p), sin(p)));
    }
}
 
extern "C" void mul_veff_with_phase_factors_gpu(int num_atoms__,
                                                int num_gvec_loc__, 
                                                hipDoubleComplex const* veff__, 
                                                int const* gvec__, 
                                                double const* atom_pos__,
                                                hipDoubleComplex* veff_a__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec_loc__, grid_t.x), num_atoms__);

    mul_veff_with_phase_factors_gpu_kernel <<<grid_b, grid_t>>>
    (
        num_gvec_loc__,
        veff__,
        gvec__,
        atom_pos__,
        veff_a__
    );
}
