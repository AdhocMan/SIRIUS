#include "hip/hip_runtime.h"
// Copyright (c) 2013-2018 Anton Kozhevnikov, Thomas Schulthess
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
//    and the following disclaimer in the documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/** \file add_pw_ekin.cu
 *
 *  \brief CUDA kernel for the hphi update.
 */

#include "../SDDK/GPU/cuda_common.hpp"
#include <hip/hip_complex.h>

__global__ void add_pw_ekin_gpu_kernel(int num_gvec__,
                                       double alpha__,
                                       double const* pw_ekin__,
                                       hipDoubleComplex const* phi__,
                                       hipDoubleComplex const* vphi__,
                                       hipDoubleComplex* hphi__)
{
    int ig = blockIdx.x * blockDim.x + threadIdx.x;
    if (ig < num_gvec__) {
        hipDoubleComplex z1 = hipCadd(vphi__[ig], make_hipDoubleComplex(alpha__ * pw_ekin__[ig] * phi__[ig].x, 
                                                                     alpha__ * pw_ekin__[ig] * phi__[ig].y));
        hphi__[ig] = hipCadd(hphi__[ig], z1);
    }
}

/// Update the hphi wave functions.
/** The following operation is performed:
 *    hphi[ig] += (alpha *  pw_ekin[ig] * phi[ig] + vphi[ig])
 */
extern "C" void add_pw_ekin_gpu(int num_gvec__,
                                double alpha__,
                                double const* pw_ekin__,
                                hipDoubleComplex const* phi__,
                                hipDoubleComplex const* vphi__,
                                hipDoubleComplex* hphi__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec__, grid_t.x));

    add_pw_ekin_gpu_kernel <<<grid_b, grid_t>>>
    (
        num_gvec__,
        alpha__,
        pw_ekin__,
        phi__,
        vphi__,
        hphi__
    );

}
