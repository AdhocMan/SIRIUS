#include "hip/hip_runtime.h"
#include "kernels_common.h"

template <int direction, bool conjugate>
__global__ void pack_unpack_z_cols_gpu_kernel
(
    hipDoubleComplex* z_cols_packed__,
    hipDoubleComplex* fft_buf__,
    int size_x__,
    int size_y__,
    int size_z__,
    int num_z_cols__,
    int const* z_columns_pos__
)
{
    int icol = blockIdx.x * blockDim.x + threadIdx.x;
    int iz = blockIdx.y;
    if (icol < num_z_cols__)
    {
        int x, y;

        if (conjugate)
        {
            x = (-z_columns_pos__[array2D_offset(0, icol, 2)] + size_x__) % size_x__;
            y = (-z_columns_pos__[array2D_offset(1, icol, 2)] + size_y__) % size_y__;
        }
        else
        {
            x = (z_columns_pos__[array2D_offset(0, icol, 2)] + size_x__) % size_x__;
            y = (z_columns_pos__[array2D_offset(1, icol, 2)] + size_y__) % size_y__;
        }
        
        /* load into buffer */
        if (direction == 1)
        {
            if (conjugate)
            {
                fft_buf__[array3D_offset(x, y, iz, size_x__, size_y__)] = hipConj(z_cols_packed__[array2D_offset(iz, icol, size_z__)]);
            }
            else
            {
                fft_buf__[array3D_offset(x, y, iz, size_x__, size_y__)] = z_cols_packed__[array2D_offset(iz, icol, size_z__)];
            }
        }
        if (direction == -1)
        {
            z_cols_packed__[array2D_offset(iz, icol, size_z__)] = fft_buf__[array3D_offset(x, y, iz, size_x__, size_y__)];
        }
    }
}

extern "C" void unpack_z_cols_gpu(hipDoubleComplex* z_cols_packed__,
                                  hipDoubleComplex* fft_buf__,
                                  int size_x__,
                                  int size_y__,
                                  int size_z__,
                                  int num_z_cols__,
                                  int const* z_columns_pos__,
                                  bool use_reduction__, 
                                  int stream_id__)
{
    hipStream_t stream = (stream_id__ == -1) ? NULL : streams[stream_id__];

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_z_cols__, grid_t.x), size_z__);

    hipMemsetAsync(fft_buf__, 0, size_x__ * size_y__ * size_z__ * sizeof(hipDoubleComplex), stream);

    pack_unpack_z_cols_gpu_kernel<1, false> <<<grid_b, grid_t, 0, stream>>>
    (
        z_cols_packed__,
        fft_buf__,
        size_x__,
        size_y__,
        size_z__,
        num_z_cols__,
        z_columns_pos__
    );
    if (use_reduction__)
    {
        pack_unpack_z_cols_gpu_kernel<1, true> <<<grid_b, grid_t, 0, stream>>>
        (
            &z_cols_packed__[size_z__],
            fft_buf__,
            size_x__,
            size_y__,
            size_z__,
            num_z_cols__ - 1,
            &z_columns_pos__[2] // * num_z_cols__]
        );
    }
}

extern "C" void pack_z_cols_gpu(hipDoubleComplex* z_cols_packed__,
                                hipDoubleComplex* fft_buf__,
                                int size_x__,
                                int size_y__,
                                int size_z__,
                                int num_z_cols__,
                                int const* z_columns_pos__,
                                int stream_id__)
{
    hipStream_t stream = (stream_id__ == -1) ? NULL : streams[stream_id__];

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_z_cols__, grid_t.x), size_z__);

    pack_unpack_z_cols_gpu_kernel<-1, false> <<<grid_b, grid_t, 0, stream>>>
    (
        z_cols_packed__,
        fft_buf__,
        size_x__,
        size_y__,
        size_z__,
        num_z_cols__,
        z_columns_pos__
    );
}



