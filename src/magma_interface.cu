// This file must be compiled with nvcc

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <magma.h>
#include <magma_z.h>
#include <magma_zbulge.h>
#include <magma_threadsetting.h>
#include "gpu_interface.h"

extern "C" void magma_init_wrapper()
{
    magma_init();
}

extern "C" void magma_finalize_wrapper()
{
    magma_finalize();
}

extern "C" void magma_zhegvdx_2stage_wrapper(int32_t matrix_size, int32_t nv, void* a, int32_t lda, void* b, 
                                             int32_t ldb, double* eval)
{
    int m;
    int info;

    int lwork = magma_zbulge_get_lq2(matrix_size, magma_get_parallel_numthreads()) + 2 * matrix_size + matrix_size * matrix_size;
    int lrwork = 1 + 5 * matrix_size + 2 * matrix_size * matrix_size;
    int liwork = 3 + 5 * matrix_size;
            
    magmaDoubleComplex* h_work = (magmaDoubleComplex*)cuda_malloc_host(lwork * sizeof(magmaDoubleComplex));
    double* rwork = (double*)cuda_malloc_host(lrwork * sizeof(double));
    
    magma_int_t *iwork;
    if ((iwork = (magma_int_t*)malloc(liwork * sizeof(magma_int_t))) == NULL)
    {
        printf("malloc failed\n");
        exit(-1);
    }
    
    double* w;
    if ((w = (double*)malloc(matrix_size * sizeof(double))) == NULL)
    {
        printf("malloc failed\n");
        exit(-1);
    }

    magma_zhegvdx_2stage(1, MagmaVec, MagmaRangeI, MagmaLower, matrix_size, (magmaDoubleComplex*)a, lda, (magmaDoubleComplex*)b, ldb, 0.0, 0.0, 
                         1, nv, &m, w, h_work, lwork, rwork, lrwork, iwork, liwork, &info);

    memcpy(eval, &w[0], nv * sizeof(double));
    
    cuda_free_host((void**)&h_work);
    cuda_free_host((void**)&rwork);
    free(iwork);
    free(w);

    if (info)
    {
        printf("magma_zhegvdx_2stage returned : %i\n", info);
        exit(-1);
    }    

    if (m != nv)
    {
        printf("Not all eigen-values are found.\n");
        exit(-1);
    }
}


